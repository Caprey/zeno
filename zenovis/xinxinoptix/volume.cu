#include "hip/hip_runtime.h"
#include "Light.h"
#include "volume.h"

#include "TraceStuff.h"
#include "zxxglslvec.h"
#include "hip/hip_math_constants.h"

// #include <hip/hip_fp16.h>
// #include "nvfunctional"
#include <nanovdb/NanoVDB.h>
#include <nanovdb/util/Ray.h>
#include <nanovdb/util/HDDA.h>
#include <nanovdb/util/SampleFromVoxels.h>

using DataTypeNVDB0 = nanovdb::Fp32;
using GridTypeNVDB0 = nanovdb::NanoGrid<DataTypeNVDB0>;

extern "C" __global__ void __intersection__volume()
{
    const auto* sbt_data = reinterpret_cast<const HitGroupData*>( optixGetSbtDataPointer() );
    const auto* grid = reinterpret_cast<const GridTypeNVDB0*>( sbt_data->vdb_grids[0] );
    if ( grid == nullptr) { return; }

    const float3 ray_orig = optixGetWorldRayOrigin(); //optixGetObjectRayOrigin();
    const float3 ray_dir  = optixGetWorldRayDirection(); //optixGetObjectRayDirection();

    auto dbox = grid->worldBBox(); //grid->indexBBox();
    float t0 = optixGetRayTmin();
    float t1 = _FLT_MAX_; //optixGetRayTmax();

    auto iray = nanovdb::Ray<float>( reinterpret_cast<const nanovdb::Vec3f&>( ray_orig ),
                                     reinterpret_cast<const nanovdb::Vec3f&>( ray_dir ), t0, t1 );
    // auto fbox = nanovdb::BBox<nanovdb::Vec3f>(nanovdb::Vec3f(dbox.min()), nanovdb::Vec3f(dbox.max()));

    if( iray.intersects( dbox, t0, t1 )) // t0 >= 0
    {
        // report the entry-point as hit-point
        //auto kind = optixGetHitKind();
        t0 = fmaxf(t0, optixGetRayTmin());

        RadiancePRD* prd = getPRD();
        prd->vol_t0 = t0;
        prd->origin_inside_vdb = (t0 == 0);

        prd->vol_t1 = t1; //min(optixGetRayTmax(), t1);
        prd->surface_inside_vdb = (optixGetRayTmax() < t1); // In case triangles were visited before vdb

        if (optixGetRayTmax() > 0) {
            optixReportIntersection(t0, 0);
        }
    } 
}

extern "C" __global__ void __closesthit__radiance_volume()
{
    RadiancePRD* prd = getPRD();
    //if(prd->test_distance) { return; }
    
    prd->countEmitted = false;
    prd->radiance = vec3(0);

    prd->_tmin_ = 0;
    prd->_mask_ = EverythingMask;

    const HitGroupData* sbt_data = reinterpret_cast<HitGroupData*>( optixGetSbtDataPointer() );

    const float3 ray_orig = optixGetWorldRayOrigin();
    const float3 ray_dir  = optixGetWorldRayDirection();

        float t0 = prd->vol_t0; // world space
        float t1 = prd->vol_t1; // world space

    RadiancePRD testPRD {};
    testPRD.vol_t1 = _FLT_MAX_;
    testPRD.test_distance = true;
    testPRD.isSS = false;


    uint16_t _mask_ = EverythingMask ^ VolumeMatMask;

    traceRadiance(params.handle, ray_orig,ray_dir, 0, _FLT_MAX_, &testPRD, _mask_);

    if(testPRD.vol_t1 < t1)
    {
        t1 = testPRD.vol_t1;
        prd->surface_inside_vdb = true;
    }

    const float t_max = fmax(0.f, t1 - t0); // world space
    float t_ele = 0;

    float3 new_orig = ray_orig; 
    float3 emitting = make_float3(0.0);
    float3 scattering = make_float3(1.0);
   
    float sigma_t = sbt_data->vol_extinction;
    float v_density = 0.0;

    VolumeOut vol_out;
    auto new_dir = ray_dir;

    auto level = sbt_data->vol_depth;
    auto step_scale = 1.0f/sigma_t;

    while(--level > 0) {
        auto prob = prd->rndf();
        t_ele -= logf(1.0f-prob) * step_scale;

        if (t_ele >= t_max) {

            if (prd->surface_inside_vdb) { // Hit other material

                prd->_mask_ = _mask_;
                prd->_tmin_ = 0;

                new_orig = ray_orig;

            } else { // Volume edge

                prd->_mask_ = EverythingMask;
                prd->_tmin_ = 1e-5f;

                new_orig = ray_orig + t1 * ray_dir;
                new_orig = rtgems::offset_ray(new_orig, ray_dir);
            }

            v_density = 0;
            break;
        } // over shoot, outside of volume

        new_orig = ray_orig + (t0+t_ele) * ray_dir;

        VolumeIn vol_in { new_orig, sigma_t, &prd->seed, reinterpret_cast<unsigned long long>(sbt_data) };

        vol_out = optixDirectCall<VolumeOut, const float4*, const VolumeIn&>( sbt_data->dc_index, sbt_data->uniforms, vol_in);
        v_density = vol_out.density;
        emitting += vol_out.emission;

        step_scale = fminf(step_scale, vol_out.step_scale) ;

        if (prd->rndf() > v_density) { // null scattering
            v_density = 0.0f; continue;
        }

        pbrt::HenyeyGreenstein hg { vol_out.anisotropy };
        float2 uu = { prd->rndf(), prd->rndf() };
        auto _ = hg.sample(-ray_dir, new_dir, uu);              
        //auto relative_prob = prob * (HIP_PI_F * 4);
        new_dir = normalize(new_dir);
        scattering = vol_out.albedo;

        if (prd->trace_denoise_normal) {
            prd->tmp_normal = normalize(-ray_dir + new_dir);
        }
        if(prd->trace_denoise_albedo) {
            prd->tmp_albedo = vol_out.albedo;
        }

        break;
    }

    prd->updateAttenuation(scattering);

    prd->origin = new_orig;
    prd->direction = new_dir;

    prd->emission = emitting;
    prd->geometryNormal = {}; //(new_dir + -ray_dir) / 2.0f;

    if (v_density == 0) {
        //prd->depth += 0;
        prd->radiance += prd->emission;
        return;
    }

    scattering = vol_out.albedo;

    RadiancePRD shadow_prd {};
    shadow_prd.seed = prd->seed;
    shadow_prd.nonThinTransHit = 0;
    shadow_prd.shadowAttanuation = vec3(1.0f);

    auto evalBxDF = [&](const float3& _wi_, const float3& _wo_, float& thisPDF, vec3 illum = vec3(1.0f)) -> float3 {

        pbrt::HenyeyGreenstein hg { vol_out.anisotropy };
        thisPDF = hg.p(_wo_, _wi_);
        return scattering * thisPDF;
    };

    DirectLighting<true>(prd, shadow_prd, new_orig, ray_dir, evalBxDF);
    
    prd->depth += 1;
    prd->radiance += prd->emission;
    
    return;
}

extern "C" __global__ void __anyhit__occlusion_volume()
{
    const float3 ray_orig = optixGetWorldRayOrigin();
    const float3 ray_dir  = optixGetWorldRayDirection();

    RadiancePRD* prd = getPRD();
    const HitGroupData* sbt_data = reinterpret_cast<HitGroupData*>( optixGetSbtDataPointer() );

    const float t0 = prd->vol_t0;
    const float t1 = prd->vol_t1;

    const float t_max = t1 - t0; // world space
          float t_ele = 0;

    float3 test_point = ray_orig; 
    float3 transmittance = make_float3(1.0f);

    const float sigma_t = sbt_data->vol_extinction;

    auto level = sbt_data->vol_depth;
    while(--level > 0) {

        auto prob = prd->rndf();
        t_ele -= log(1.0f-prob) / (sigma_t);

        test_point = ray_orig + (t0+t_ele) * ray_dir;

        if (t_ele >= t_max) {
            break;
        } // over shoot, outside of volume

        VolumeIn vol_in { test_point, sigma_t, &prd->seed, reinterpret_cast<unsigned long long>(sbt_data) };
        VolumeOut vol_out = optixDirectCall<VolumeOut, const float4*, const VolumeIn&>( sbt_data->dc_index, sbt_data->uniforms, vol_in );

        const auto v_density = vol_out.density;

        auto prob_scatter = clamp(v_density, 0.0f, 1.0f);
        auto prob_nulling = 1.0f - prob_scatter;

        pbrt::HenyeyGreenstein hg { vol_out.anisotropy };
        auto prob_continue = hg.p(-ray_dir, ray_dir) * prob_scatter;
        prob_continue = clamp(prob_continue, 0.0, 1.0f);
        //printf("prob_continue %f \n", prob_continue);

        auto tr = transmittance * prob_nulling;
        tr += transmittance * prob_continue * vol_out.albedo;
        
        transmittance = clamp(tr, 0.0, 1.0f);

        auto avg = dot(transmittance, make_float3(1.0f/3.0f));
        if (avg < 0.1f) {
            float q = fmax(0.05f, 1 - avg);
            if (prd->rndf() < q) { 
                transmittance = vec3(0);
                break; 
            } else {
                transmittance /= 1-q;
            }
        }
    }

    prd->shadowAttanuation *= transmittance;
    optixIgnoreIntersection();
    //prd->origin = ray_orig;
    //prd->direction = ray_dir;
    return;
}