#include "hip/hip_runtime.h"
// #include "zensim/container/Vector.hpp"
// #include "zensim/geometry/VdbLevelSet.h"
#include "zensim/cuda/Cuda.h"
#include "zensim/cuda/memory/MemOps.hpp"
#include "zensim/geometry/AdaptiveGrid.hpp"
#include "zensim/memory/Allocator.h"
#include <cassert>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hiprtc.h>
#include <zeno/types/DictObject.h>
#include <zeno/types/ListObject.h>
#include <zeno/types/NumericObject.h>
#include <zeno/types/PrimitiveObject.h>
#include <zeno/types/StringObject.h>
#include <zeno/zeno.h>
namespace zeno {

__global__ void test(int *a) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    printf("[%d]: %d\n", id, a[id]);
}

struct ZSCULinkTest : INode {
    void apply() override {

        {
            zs::AdaptiveGrid<3, float, 3, 4, 5> ag;
            using TT = RM_CVREF_T(ag);
            fmt::print("adaptive grid type: {}\n", zs::get_var_type_str(ag));
            fmt::print("tile bits: {}\n", zs::get_type_str<TT::tile_bits_type>());
            fmt::print("hierarchy bits: {}\n", zs::get_type_str<TT::hierarchy_bits_type>());

            fmt::print("num total blocks: {}\n", ag.numTotalBlocks());
            auto hag = ag.clone({zs::memsrc_e::host, -1});
        }

        constexpr int n = 100;
        // hipInit(0);
        (void)zs::Cuda::instance();
        puts("1");
        int *a = nullptr;
        // hipMalloc((void **)&a, n * sizeof(int));
        a = (int *)zs::allocate(zs::mem_um, n * sizeof(int), sizeof(int));
        puts("2");

#if 1
        std::vector<int> ha(n);
#else
        zs::Vector<int> ha{n, zs::memsrc_e::host, -1};
#endif
        for (int i = 0; i != n; ++i)
            ha[i] = i;
        puts("3");
        hipMemcpy(a, ha.data(), n * sizeof(int), hipMemcpyHostToDevice);
        test<<<1, n>>>(a);
        hipDeviceSynchronize();

        puts("4");
        // hipFree(a);
        // zs::deallocate(zs::mem_um, a, );
        zs::raw_memory_resource<zs::um_mem_tag>::instance().deallocate(a, n * sizeof(int));
        puts("5");

        hiprtcProgram prog;
        hiprtcCreateProgram(&prog, "", "ahh", 0, NULL, NULL);

        printf("done!\n");
        getchar();
    }
};

ZENDEFNODE(ZSCULinkTest, {
                             {},
                             {},
                             {},
                             {"ZPCTest"},
                         });

} // namespace zeno
