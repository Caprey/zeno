#include "hip/hip_runtime.h"
#include "Structures.hpp"
#include "zensim/cuda/execution/ExecutionPolicy.cuh"
#include <cassert>
#include <cstdlib>
#include <cstring>
#include <random>
#include <zeno/types/DummyObject.h>
#include <zeno/types/ListObject.h>
#include <zeno/types/NumericObject.h>
#include <zeno/types/PrimitiveObject.h>
#include <zeno/types/UserData.h>
#include <zeno/utils/log.h>
#include <zeno/utils/parallel_reduce.h>
#include <zeno/utils/vec.h>
#include <zeno/zeno.h>

namespace zeno {

/// utilities
constexpr std::size_t count_warps(std::size_t n) noexcept {
    return (n + 31) / 32;
}
constexpr int warp_index(int n) noexcept {
    return n / 32;
}
constexpr auto warp_mask(int i, int n) noexcept {
    int k = n % 32;
    const int tail = n - k;
    if (i < tail)
        return zs::make_tuple(0xFFFFFFFFu, 32);
    return zs::make_tuple(((unsigned)(1ull << k) - 1), k);
}

template <typename T, typename Op>
__forceinline__ __device__ void reduce_to(int i, int n, T val, T &dst, Op op) {
    auto [mask, numValid] = warp_mask(i, n);
    __syncwarp(mask);
    auto locid = threadIdx.x & 31;
    for (int stride = 1; stride < 32; stride <<= 1) {
        auto tmp = __shfl_down_sync(mask, val, stride);
        if (locid + stride < numValid)
            val = op(val, tmp);
    }
    if (locid == 0)
        dst = val;
}

template <typename TransOp, typename ReduceOp>
float prim_reduce(typename ZenoParticles::particles_t &verts, float e, TransOp top, ReduceOp rop,
                  std::string attrToReduce) {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;
    using T = typename ZenoParticles::particles_t::value_type;
    auto nchn = verts.getPropertySize(attrToReduce);
    auto offset = verts.getPropertyOffset(attrToReduce);
    const auto nwarps = count_warps(verts.size());

    auto cudaPol = cuda_exec();

    Vector<float> res{verts.get_allocator(), nwarps};
    // cudaPol(res, [e] ZS_LAMBDA(auto &v) { v = e; });
    cudaPol(range(verts.size()), [res = proxy<space>(res), verts = proxy<space>({}, verts), offset, nwarps, nchn, top,
                                  rop] ZS_LAMBDA(int i) mutable {
        auto [mask, numValid] = warp_mask(i, nwarps);
        float v = top(verts(offset, i));
        while (--nchn) {
            v = rop(top(verts(offset++, i)), v);
        }
        reduce_to(i, nwarps, v, res[i / 32], rop);
    });

    Vector<float> ret{res.get_allocator(), 1};
    zs::reduce(cudaPol, std::begin(res), std::end(res), std::begin(ret), e, rop);
    return ret.getVal();
}

struct ZSPrimitiveReduction : zeno::INode {
    struct pass_on {
        template <typename T>
        constexpr T operator()(T v) const noexcept {
            return v;
        }
    };
    struct getabs {
        template <typename T>
        constexpr T operator()(T v) const noexcept {
            return zs::abs(v);
        }
    };
    virtual void apply() override {
        using namespace zs;
        auto prim = get_input<ZenoParticles>("ZSParticles");
        auto &verts = prim->getParticles();
        auto attrToReduce = get_input2<std::string>("attr");
        if (attrToReduce == "pos")
            attrToReduce = "x";
        if (attrToReduce == "vel")
            attrToReduce = "v";

        if (!verts.hasProperty(attrToReduce))
            throw std::runtime_error(fmt::format("verts do not have property [{}]\n", attrToReduce));

        auto opStr = get_input2<std::string>("op");
        zeno::NumericValue result;
        if (opStr == "avg") {
            result = prim_reduce(verts, 0, pass_on{}, std::plus<float>{}, attrToReduce) / verts.size();
        } else if (opStr == "max") {
            result = prim_reduce(verts, limits<float>::lowest(), pass_on{}, getmax<float>{}, attrToReduce);
        } else if (opStr == "min") {
            result = prim_reduce(verts, limits<float>::max(), pass_on{}, getmin<float>{}, attrToReduce);
        } else if (opStr == "absmax") {
            result = prim_reduce(verts, 0, getabs{}, getmax<float>{}, attrToReduce);
        }

        auto out = std::make_shared<zeno::NumericObject>();
        out->set(result);
        set_output("result", std::move(out));
    }
};
ZENDEFNODE(ZSPrimitiveReduction, {/* inputs: */ {
                                      "ZSParticles",
                                      {"string", "attr", "pos"},
                                      {"enum avg max min absmax", "op", "avg"},
                                  },
                                  /* outputs: */
                                  {
                                      "result",
                                  },
                                  /* params: */
                                  {},
                                  /* category: */
                                  {
                                      "primitive",
                                  }});

struct ZSGetUserData : zeno::INode {
    virtual void apply() override {
        auto object = get_input<ZenoParticles>("object");
        auto key = get_param<std::string>("key");
        auto hasValue = object->zsUserData().has(key);
        auto data = hasValue ? object->zsUserData().get(key) : std::make_shared<DummyObject>();
        set_output2("hasValue", hasValue);
        set_output("data", std::move(data));
    }
};

ZENDEFNODE(ZSGetUserData, {
                              {"object"},
                              {"data", {"bool", "hasValue"}},
                              {{"string", "key", ""}},
                              {"lifecycle"},
                          });

struct ColoringSelected : INode {
    using tiles_t = typename ZenoParticles::particles_t;
    void markBoundaryVerts(zs::CudaExecutionPolicy &pol, ZenoParticles *prim) {
        using namespace zs;
        auto &vtemp = prim->getParticles();
        vtemp.append_channels(pol, std::vector<zs::PropertyTag>{{"on_boundary", 1}});
        auto markIter = vtemp.begin("on_boundary", dim_c<1>, int_c);
        auto markIterEnd = vtemp.end("on_boundary", dim_c<1>, int_c);
        pol(detail::iter_range(markIter, markIterEnd), [] ZS_LAMBDA(auto &mark) mutable { mark = 0; });

        if (prim->category == ZenoParticles::curve) {
            auto &eles = prim->getQuadraturePoints();
            mark_surface_boundary_verts(pol, eles, wrapv<2>{}, markIter, (size_t)0);
        } else if (prim->category == ZenoParticles::surface) {
            auto &eles = prim->getQuadraturePoints();
            mark_surface_boundary_verts(pol, eles, wrapv<3>{}, markIter, (size_t)0);
        } else if (prim->category == ZenoParticles::tet) {
            auto &surf = (*prim)[ZenoParticles::s_surfTriTag];
            mark_surface_boundary_verts(pol, surf, wrapv<3>{}, markIter, (size_t)0);
        }
    }
    template <typename LsView>
    void markVerts(zs::CudaExecutionPolicy &cudaPol, zs::SmallString tag, ZenoParticles *zsprim, LsView lsv,
                   bool boundaryWise) {
        using namespace zs;
        constexpr auto space = execspace_e::cuda;
        auto &vtemp = zsprim->getParticles();
        auto numVerts = vtemp.size();
        cudaPol(range(numVerts), [vtemp = proxy<space>({}, vtemp), tag, lsv, boundaryWise] ZS_LAMBDA(int i) mutable {
            if (boundaryWise && vtemp.hasProperty("on_boundary"))
                if (vtemp("on_boundary", i, int_c) == 0) // only operate on verts on boundary
                    return;
            auto x = vtemp.pack(dim_c<3>, "x", i);
            if (lsv.getSignedDistance(x) < 0) {
                vtemp(tag, i) = 1.f;
            }
        });
    }
    void apply() override {
        using namespace zs;
        constexpr auto space = execspace_e::cuda;
        auto zsprim = get_input<ZenoParticles>("ZSParticles");

        auto cudaPol = zs::cuda_exec().sync(true);

        auto zsls = get_input<ZenoLevelSet>("ZSLevelSet");
        bool boundaryWise = get_input2<bool>("boundary_wise");
        auto &vtemp = zsprim->getParticles();
        if (boundaryWise || vtemp.hasProperty("on_boundary"))
            markBoundaryVerts(cudaPol, zsprim.get());

        auto tag = get_input2<std::string>("markTag");
        vtemp.append_channels(cudaPol, std::vector<zs::PropertyTag>{{tag, 1}});
        cudaPol(range(vtemp, tag), [] ZS_LAMBDA(auto &mark) mutable { mark = 0; });

        match([&](const auto &ls) {
            using basic_ls_t = typename ZenoLevelSet::basic_ls_t;
            using const_sdf_vel_ls_t = typename ZenoLevelSet::const_sdf_vel_ls_t;
            using const_transition_ls_t = typename ZenoLevelSet::const_transition_ls_t;
            if constexpr (is_same_v<RM_CVREF_T(ls), basic_ls_t>) {
                match([&](const auto &lsPtr) {
                    auto lsv = get_level_set_view<execspace_e::cuda>(lsPtr);
                    markVerts(cudaPol, tag, zsprim.get(), lsv, boundaryWise);
                })(ls._ls);
            } else if constexpr (is_same_v<RM_CVREF_T(ls), const_sdf_vel_ls_t>) {
                match([&](auto lsv) { markVerts(cudaPol, tag, zsprim.get(), SdfVelFieldView{lsv}, boundaryWise); })(
                    ls.template getView<execspace_e::cuda>());
            } else if constexpr (is_same_v<RM_CVREF_T(ls), const_transition_ls_t>) {
                match([&](auto fieldPair) {
                    auto &fvSrc = zs::get<0>(fieldPair);
                    auto &fvDst = zs::get<1>(fieldPair);
                    markVerts(
                        cudaPol, tag, zsprim.get(),
                        TransitionLevelSetView{SdfVelFieldView{fvSrc}, SdfVelFieldView{fvDst}, ls._stepDt, ls._alpha},
                        boundaryWise);
                })(ls.template getView<zs::execspace_e::cuda>());
            }
        })(zsls->getLevelSet());

        set_output("ZSParticles", zsprim);
    }
};

ZENDEFNODE(ColoringSelected, {{
                                  "ZSParticles",
                                  "ZSLevelSet",
                                  {"bool", "boundary_wise", "0"},
                                  {"string", "markTag", "selected"},
                              },
                              {"ZSParticles"},
                              {},
                              {"geom"}});

} // namespace zeno