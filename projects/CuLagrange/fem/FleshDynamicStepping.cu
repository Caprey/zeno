#include "hip/hip_runtime.h"
#include "Structures.hpp"
#include "zensim/Logger.hpp"
#include "zensim/cuda/execution/ExecutionPolicy.cuh"
#include "zensim/omp/execution/ExecutionPolicy.hpp"
#include "zensim/geometry/PoissonDisk.hpp"
#include "zensim/geometry/VdbLevelSet.h"
#include "zensim/geometry/VdbSampler.h"
#include "zensim/io/MeshIO.hpp"
#include "zensim/math/bit/Bits.h"
#include "zensim/types/Property.h"
#include <atomic>
#include <zeno/VDBGrid.h>
#include <zeno/types/ListObject.h>
#include <zeno/types/NumericObject.h>
#include <zeno/types/PrimitiveObject.h>
#include <zeno/types/StringObject.h>

#include "../geometry/linear_system/mfcg.hpp"

#include "../geometry/kernel/calculate_facet_normal.hpp"
#include "../geometry/kernel/topology.hpp"
#include "../geometry/kernel/compute_characteristic_length.hpp"
#include "../geometry/kernel/calculate_bisector_normal.hpp"

#include "../geometry/kernel/tiled_vector_ops.hpp"
#include "../geometry/kernel/geo_math.hpp"

#include "../geometry/kernel/calculate_edge_normal.hpp"

#include "zensim/container/Bvh.hpp"
#include "zensim/container/Bvs.hpp"
#include "zensim/container/Bvtt.hpp"

#include "collision_energy/vertex_face_sqrt_collision.hpp"
#include "collision_energy/vertex_face_collision.hpp"
// #include "collision_energy/edge_edge_sqrt_collision.hpp"
// #include "collision_energy/edge_edge_collision.hpp"

#include "collision_energy/evaluate_collision.hpp"
#include "../geometry/kernel/intersection.hpp"

#include "zensim/math/matrix/SparseMatrix.hpp"

namespace zeno {

#define MAX_FP_COLLISION_PAIRS 4

#define USE_SPARSE_MATRIX


template <typename SpmatT, typename VecTM, typename VecTI,
          zs::enable_if_all<VecTM::dim == 2, VecTM::template range_t<0>::value == VecTM::template range_t<1>::value,
                            VecTI::dim == 1, VecTI::extent * 3 == VecTM::template range_t<0>::value> = 0>
__forceinline__ __device__ void
update_hessian(cooperative_groups::thread_block_tile<8, cooperative_groups::thread_block> &tile, SpmatT &spmat,
               const VecTI &inds, const VecTM &hess) {
    using namespace zs;
    constexpr int codim = VecTI::extent;
    using mat3 = typename SpmatT::value_type;
    const auto nnz = spmat.nnz();
    const int cap = __popc(tile.ballot(1)); // assume active pattern 0...001111 [15, 14, ..., 0]
    auto laneId = tile.thread_rank();
#pragma unroll
    for (int i = 0; i != codim; ++i) {
        auto subOffsetI = i * 3;
        auto row = inds[i];
        // diagonal
        auto loc = spmat._ptrs[row];
        auto &mat = const_cast<mat3 &>(spmat._vals[loc]);

        for (int d = laneId; d < 9; d += cap) {
            atomic_add(exec_cuda, &mat(d / 3, d % 3), hess(subOffsetI + d / 3, subOffsetI + d % 3));
        }
        // non-diagonal
        for (int j = i + 1; j < codim; ++j) {
            auto subOffsetJ = j * 3;
            auto col = inds[j];
            if (row < col) {
                auto loc = spmat.locate(row, col, zs::true_c);
                auto &mat = const_cast<mat3 &>(spmat._vals[loc]);
                for (int d = laneId; d < 9; d += cap)
                    atomic_add(exec_cuda, &mat.val(d), hess(subOffsetI + d / 3, subOffsetJ + d % 3));
            } else {
                auto loc = spmat.locate(col, row, zs::true_c);
                auto &mat = const_cast<mat3 &>(spmat._vals[loc]);
                for (int d = laneId; d < 9; d += cap)
                    atomic_add(exec_cuda, &mat.val(d), hess(subOffsetI + d % 3, subOffsetJ + d / 3));
            }
        }
    }
}
template <typename T, zs::enable_if_t<std::is_fundamental_v<T>> = 0>
__forceinline__ __device__ T tile_shfl(cooperative_groups::thread_block_tile<8, cooperative_groups::thread_block> &tile,
                                       T var, int srcLane) {
    return tile.shfl(var, srcLane);
}
template <typename VecT, zs::enable_if_t<zs::is_vec<VecT>::value> = 0>
__forceinline__ __device__ VecT tile_shfl(
    cooperative_groups::thread_block_tile<8, cooperative_groups::thread_block> &tile, const VecT &var, int srcLane) {
    VecT ret{};
    for (typename VecT::index_type i = 0; i != VecT::extent; ++i)
        ret.val(i) = tile_shfl(tile, var.val(i), srcLane);
    return ret;
}
template <typename SpmatT, typename VecTM, typename VecTI,
          zs::enable_if_all<VecTM::dim == 2, VecTM::template range_t<0>::value == VecTM::template range_t<1>::value,
                            VecTI::dim == 1, VecTI::extent * 3 == VecTM::template range_t<0>::value> = 0>
__forceinline__ __device__ void update_hessian(SpmatT &spmat, const VecTI &inds, const VecTM &hess,
                                               bool has_work = true) {
    using namespace zs;
    // constexpr int codim = VecTI::extent;
    auto tile = cg::tiled_partition<8>(cg::this_thread_block());

    u32 work_queue = tile.ballot(has_work);
    while (work_queue) {
        auto cur_rank = __ffs(work_queue) - 1;
        auto cur_work = tile_shfl(tile, hess, cur_rank);
        auto cur_index = tile.shfl(inds, cur_rank); // gather index as well
        update_hessian(tile, spmat, cur_index, cur_work);

        if (tile.thread_rank() == cur_rank)
            has_work = false;
        work_queue = tile.ballot(has_work);
    }
    return;
}

struct FleshDynamicStepping : INode {

    using T = float;
    using Ti = int;
    using dtiles_t = zs::TileVector<T,32>;
    using tiles_t = typename ZenoParticles::particles_t;
    using vec2 = zs::vec<T,2>;
    using vec3 = zs::vec<T, 3>;
    using mat3 = zs::vec<T, 3, 3>;
    using mat9 = zs::vec<T,9,9>;
    using mat12 = zs::vec<T,12,12>;

    using bvh_t = zs::LBvh<3,int,T>;
    using bv_t = zs::AABBBox<3, T>;

    using pair3_t = zs::vec<Ti,3>;
    using pair4_t = zs::vec<Ti,4>;

    using spmat_t = zs::SparseMatrix<mat3, true>;

    // currently only backward euler integrator is supported
    // topology evaluation should be called before applying this node
    struct FEMDynamicSteppingSystem {
        template <typename Model>
        void computeCollisionEnergy(zs::CudaExecutionPolicy& cudaPol,const Model& model,
                dtiles_t& vtemp,
                dtiles_t& etemp,
                dtiles_t& sttemp,
                dtiles_t& setemp,
                dtiles_t& ee_buffer,
                dtiles_t& fe_buffer) {
            using namespace zs;
            constexpr auto space = execspace_e::cuda;

            T lambda = model.lam;
            T mu = model.mu;
        }


        void findInversion(zs::CudaExecutionPolicy& cudaPol,dtiles_t& vtemp,dtiles_t& etemp) {
            using namespace zs;
            constexpr auto space = execspace_e::cuda;
            TILEVEC_OPS::fill(cudaPol,vtemp,"is_inverted",(T)0.0);  
            TILEVEC_OPS::fill(cudaPol,etemp,"is_inverted",(T)0.0);  
            cudaPol(zs::range(eles.size()),
                [vtemp = proxy<space>({},vtemp),
                        quads = proxy<space>({},eles),
                        etemp = proxy<space>({},etemp)] ZS_LAMBDA(int ei) mutable {
                    auto DmInv = quads.template pack<3,3>("IB",ei);
                    auto inds = quads.template pack<4>("inds",ei).reinterpret_bits(int_c);
                    vec3 x1[4] = {vtemp.template pack<3>("xn", inds[0]),
                            vtemp.template pack<3>("xn", inds[1]),
                            vtemp.template pack<3>("xn", inds[2]),
                            vtemp.template pack<3>("xn", inds[3])};   

                    mat3 F{};
                    {
                        auto x1x0 = x1[1] - x1[0];
                        auto x2x0 = x1[2] - x1[0];
                        auto x3x0 = x1[3] - x1[0];
                        auto Ds = mat3{x1x0[0], x2x0[0], x3x0[0], x1x0[1], x2x0[1],
                                        x3x0[1], x1x0[2], x2x0[2], x3x0[2]};
                        F = Ds * DmInv;
                    } 
                    if(zs::determinant(F) < 0.0){
                        // for(int i = 0;i < 4;++i)
                        //     vtemp("is_inverted",inds[i]) = reinterpret_bits<T>((int)1);   
                        etemp("is_inverted",ei) = (T)1.0;   
                    }
                    else {
                        etemp("is_inverted",ei) = (T)0.0;   
                    }               
            });
            cudaPol(zs::range(eles.size()),
                [vtemp = proxy<space>({},vtemp),
                        quads = proxy<space>({},eles),
                        etemp = proxy<space>({},etemp)] ZS_LAMBDA(int ei) mutable {
                auto inds = quads.template pack<4>("inds",ei).reinterpret_bits(int_c);
                auto is_inverted = etemp("is_inverted",ei) > (T)0.5;  
                if(is_inverted)
                    for(int i = 0;i != 4;++i){
                        vtemp("is_inverted",inds[i]) = (T)1.0;     
                    }       
            });
        }


        void accumInversion(zs::CudaExecutionPolicy& cudaPol,dtiles_t& vtemp,dtiles_t& etemp) {
            using namespace zs;
            constexpr auto space = execspace_e::cuda;
            cudaPol(zs::range(eles.size()),
                [vtemp = proxy<space>({},vtemp),
                        quads = proxy<space>({},eles),
                        etemp = proxy<space>({},etemp)] ZS_LAMBDA(int ei) mutable {
                    auto DmInv = quads.template pack<3,3>("IB",ei);
                    auto inds = quads.template pack<4>("inds",ei).reinterpret_bits(int_c);
                    vec3 x1[4] = {vtemp.template pack<3>("xn", inds[0]),
                            vtemp.template pack<3>("xn", inds[1]),
                            vtemp.template pack<3>("xn", inds[2]),
                            vtemp.template pack<3>("xn", inds[3])};   

                    mat3 F{};
                    {
                        auto x1x0 = x1[1] - x1[0];
                        auto x2x0 = x1[2] - x1[0];
                        auto x3x0 = x1[3] - x1[0];
                        auto Ds = mat3{x1x0[0], x2x0[0], x3x0[0], x1x0[1], x2x0[1],
                                        x3x0[1], x1x0[2], x2x0[2], x3x0[2]};
                        F = Ds * DmInv;
                    } 
                    if(zs::determinant(F) < 0.0){
                        // for(int i = 0;i < 4;++i)
                        //     vtemp("is_inverted",inds[i]) = reinterpret_bits<T>((int)1);   
                        etemp("is_inverted",ei) = (T)1.0;   
                    }
                    // else {
                    //     etemp("is_inverted",ei) = reinterpret_bits<T>((int)0);   
                    // }               
            });
            cudaPol(zs::range(eles.size()),
                [vtemp = proxy<space>({},vtemp),
                        quads = proxy<space>({},eles),
                        etemp = proxy<space>({},etemp)] ZS_LAMBDA(int ei) mutable {
                auto inds = quads.template pack<4>("inds",ei).reinterpret_bits(int_c);
                auto is_inverted = etemp("is_inverted",ei) > (T)0.5;  
                if(is_inverted)
                    for(int i = 0;i != 4;++i){
                        vtemp("is_inverted",inds[i]) = (T)1.0;     
                    }       
            });
        }


        void computePositionConstraintGradientAndHessian(zs::CudaExecutionPolicy& cudaPol,
            dtiles_t& vtemp,
            const std::string& binderTag,
            const std::string& thicknessTag,
            const std::string& inversionTag,
            const dtiles_t& kverts,
            dtiles_t& gh_buffer) {
                using namespace zs;
                constexpr auto space = execspace_e::cuda;
                int max_nm_binders = tris.getPropertySize(binderTag);
                // printf("max_nm_binders = %d\n",max_nm_binders);

                cudaPol(zs::range(tris.size()),
                    [vtemp = proxy<space>({},vtemp),
                        verts = proxy<space>({},verts),
                        eles = proxy<space>({},eles),
                        binderTag = zs::SmallString(binderTag),
                        thicknessTag = zs::SmallString(thicknessTag),
                        inversionTag = zs::SmallString(inversionTag),
                        tris = proxy<space>({},tris),
                        kverts = proxy<space>({},kverts),
                        binderStiffness = binderStiffness,
                        max_nm_binders = max_nm_binders,
                        gh_buffer = proxy<space>({},gh_buffer)] ZS_LAMBDA(int ti) mutable {
                    int nm_binders = 0;

                    for(int i = 0;i != max_nm_binders;++i){
                        auto idx = reinterpret_bits<int>(tris(binderTag,i,ti));
                        if(idx < 0)
                            break;
                        ++nm_binders;
                    }

                    // printf("binder_ids[%d] : %d : %d %d %d %d\n",ti,nm_binders,
                    //     reinterpret_bits<int>(tris(binderTag,0,ti)),
                    //     reinterpret_bits<int>(tris(binderTag,1,ti)),
                    //     reinterpret_bits<int>(tris(binderTag,2,ti)),
                    //     reinterpret_bits<int>(tris(binderTag,3,ti)));

                    if(nm_binders == 0)
                        return;
                    auto tri = tris.pack(dim_c<3>,"inds",ti).reinterpret_bits(int_c);
                    if(verts.hasProperty("binder_fail"))
                        for(int i = 0;i != 3;++i)
                            if(verts("binder_fail",tri[i]) > (T)0.5)
                                return;
                    auto binder_weakness_param = (T)1.0;
                    // for(int i = 0;i != 3;++i)
                    //     if(vtemp("is_inverted",tri[i]) > (T)0.5)
                    //         return;

                    auto ei = reinterpret_bits<int>(tris("ft_inds",ti));
                    auto tet = eles.pack(dim_c<4>,"inds",ei).reinterpret_bits(int_c);


                    auto mu = eles("mu",ei);
                    auto lam = eles("lam",ei);
                    // auto vole = tris("vol",ti);
                    vec3 cp[4] = {};

                    cp[1] = vtemp.pack(dim_c<3>,"xn",tri[0]);
                    cp[2] = vtemp.pack(dim_c<3>,"xn",tri[1]);
                    cp[3] = vtemp.pack(dim_c<3>,"xn",tri[2]);


                    auto inds_reorder = zs::vec<int,3>::zeros();
                    for(int i = 0;i != 3;++i){
                        auto idx = tri[i];
                        for(int j = 0;j != 4;++j)
                            if(idx == tet[j])
                                inds_reorder[i] = j;
                    }

                    for(int i = 0;i != nm_binders;++i) {
                        auto idx = reinterpret_bits<int>(tris(binderTag,i,ti));
                        
                        auto ceps = tris(thicknessTag,i,ti);
                        auto from_inside = tris(inversionTag,i,ti) > (T)0.0;
                        if(kverts.hasProperty("b_fail"))
                            if(kverts("b_fail",idx) > (T)0.5)
                                continue;

                        if(idx >= kverts.size()){
                            printf("kverts buffer overflow %d >= %d\n",idx,kverts.size());
                        }

                        cp[0] = kverts.pack(dim_c<3>,"x",idx);
                        auto kstiffness = (T)1.0;
                        if(kverts.hasProperty("binderStiffness"))
                            kstiffness = kverts("binderStiffness",idx);
                        auto alpha = binderStiffness * binder_weakness_param * kstiffness;
                        auto beta = (T)1.0/(T)nm_binders;
                        auto cgrad = -alpha * beta * VERTEX_FACE_SQRT_COLLISION::gradient(cp,mu,lam,ceps,from_inside);
                        auto cH = alpha * beta * VERTEX_FACE_SQRT_COLLISION::hessian(cp,mu,lam,ceps,from_inside);

                        // printf("cgrad : %f cH %f params: %f %f %f %f\n",cgrad.norm(),cH.norm(),
                        //         (float)kstiffness,
                        //         (float)binderStiffness,
                        //         (float)binder_weakness_param,
                        //         (float)alpha);

                        // if(isnan(cH.norm())) {
                        //     printf("nan CH detected at Binder : %d from inside %d and ceps = \n",ti,from_inside,(float)ceps);
                        //     printf("cp : \n%f %f %f\n%f %f %f\n%f %f %f\n%f %f %f\n",
                        //         (float)cp[0][0],(float)cp[0][1],(float)cp[0][2],
                        //         (float)cp[1][0],(float)cp[1][1],(float)cp[1][2],
                        //         (float)cp[2][0],(float)cp[2][1],(float)cp[2][2],
                        //         (float)cp[3][0],(float)cp[3][1],(float)cp[3][2]);
                        // }

                        for(int i = 3;i != 12;++i){
                            int d0 = i % 3;
                            int row = inds_reorder[i/3 - 1]*3 + d0;
                            atomic_add(exec_cuda,&gh_buffer("grad",row,ei),cgrad[i]);
                            for(int j = 3;j != 12;++j){
                                int d1 = j % 3;
                                int col = inds_reorder[j/3 - 1]*3 + d1;
                                if(row >= 12 || col >= 12){
                                    printf("invalid row = %d and col = %d %d %d detected %d %d %d\n",row,col,i/3,j/3,
                                        inds_reorder[0],
                                        inds_reorder[1],
                                        inds_reorder[2]);
                                }
                                atomic_add(exec_cuda,&gh_buffer("H",row*12 + col,ei),cH(i,j));
                            }                    
                        }                        
                    }
                });
        }

        template <typename Model>
        void computeCollisionGradientAndHessian(zs::CudaExecutionPolicy& cudaPol,const Model& model,
                            dtiles_t& vtemp,
                            dtiles_t& etemp,
                            dtiles_t& sttemp,
                            dtiles_t& setemp,
                            // dtiles_t& ee_buffer,
                            dtiles_t& fp_buffer,
                            dtiles_t& kverts,
                            dtiles_t& kc_buffer,
                            dtiles_t& gh_buffer,
                            T kd_theta = (T)0.0,
                            bool explicit_collision = false,
                            bool neglect_inverted = true) {
            using namespace zs;
            constexpr auto space = execspace_e::cuda;

            int offset = eles.size();

            T lambda = model.lam;
            T mu = model.mu; 

            // auto stBvh = bvh_t{};
            // auto bvs = retrieve_bounding_volumes(cudaPol,vtemp,tris,wrapv<3>{},(T)0.0,"xn");
            // stBvh.build(cudaPol,bvs);
            // auto avgl = compute_average_edge_length(cudaPol,vtemp,"xn",tris);
            // auto bvh_thickness = 5 * avgl;            
            // if(!calculate_facet_normal(cudaPol,vtemp,"xn",tris,sttemp,"nrm")){
            //     throw std::runtime_error("fail updating facet normal");
            // }       
            // if(!COLLISION_UTILS::calculate_cell_bisector_normal(cudaPol,
            //     vtemp,"xn",
            //     lines,
            //     tris,
            //     sttemp,"nrm",
            //     setemp,"nrm")){
            //         throw std::runtime_error("fail calculate cell bisector normal");
            // }    


            COLLISION_UTILS::do_facet_point_collision_detection<MAX_FP_COLLISION_PAIRS>(cudaPol,
                vtemp,"xn",
                points,
                lines,
                tris,
                sttemp,
                setemp,
                fp_buffer,
                in_collisionEps,out_collisionEps);

            COLLISION_UTILS::evaluate_fp_collision_grad_and_hessian(cudaPol,
                vtemp,"xn","vn",dt,
                fp_buffer,
                gh_buffer,offset,
                in_collisionEps,out_collisionEps,
                (T)collisionStiffness,
                (T)mu,(T)lambda,(T)kd_theta);
            


            // COLLISION_UTILS::do_kinematic_point_collision_detection<MAX_FP_COLLISION_PAIRS>(cudaPol,
            //     vtemp,"xn",
            //     points,
            //     lines,
            //     tris,
            //     setemp,
            //     sttemp,
            //     kverts,
            //     kc_buffer,
            //     (T)kine_in_collisionEps,(T)kine_out_collisionEps,false);

            // offset = 0;

            // COLLISION_UTILS::evaluate_kinematic_fp_collision_grad_and_hessian(cudaPol,
            //     eles,
            //     vtemp,"xn","vn",dt,
            //     tris,
            //     kverts,
            //     kc_buffer,
            //     gh_buffer,offset,
            //     (T)kine_in_collisionEps,(T)kine_out_collisionEps,
            //     (T)kineCollisionStiffness,
            //     (T)mu,(T)lambda,(T)kd_theta);


            // adding collision damping on self collision
            // int offset = eles.size() + b_verts.size();
            // cudaPol(zs::range(fp_buffer.size() + kc_buffer.size()),
            //     [vtemp = proxy<space>({},vtemp),
            //         gh_buffer = proxy<space>({},gh_buffer),offset,kd_theta] ZS_LAMBDA(int ci) mutable {
            //     auto inds = gh_buffer.pack(dim_c<4>,"inds",ci).reinterpret_bits(int_c);
            //     for(int i = 0;i != 4;++i)
            //         if(inds[i] < 0)
            //             return;
            //     vec3 vs[4] = {};
            //     for(int i = 0;i = 4;++i)
            //         vs[i] = vtemp.pack(dim_c<3>,"vn",inds[i]);
            //     auto H = gh_buffer.pack(dim_c<12*12>,"H",ci);
            //     gh_buffer.tuple(dim_c<12*12>,"H",ci) = H;
            // });
        

        }

        void computePlaneConstraintGradientAndHessian2(zs::CudaExecutionPolicy& cudaPol,
                            const dtiles_t& vtemp,
                            const dtiles_t& sttemp,
                            const dtiles_t& kverts,
                            const dtiles_t& ktris,
                            const std::string& planeConsBaryTag,
                            const std::string& planeConsIDTag,
                            dtiles_t& nodal_gh_buffer,
                            dtiles_t& tris_gh_buffer,
                            T cnorm,bool use_sticky_condition) {
            using namespace zs;
            constexpr auto space = execspace_e::cuda;

            cudaPol(zs::range(verts.size()),[
                    verts = proxy<space>({},verts),
                    vtemp = proxy<space>({},vtemp),
                    kverts = proxy<space>({},kverts),
                    ktris = proxy<space>({},ktris),
                    planeConsBaryTag = zs::SmallString(planeConsBaryTag),
                    planeConsIDTag = zs::SmallString(planeConsIDTag),
                    kine_out_collisionEps = kine_out_collisionEps,
                    plane_constraint_stiffness = plane_constraint_stiffness,
                    use_sticky_condition = use_sticky_condition,
                    nodal_gh_buffer = proxy<space>({},nodal_gh_buffer)] ZS_LAMBDA(int vi) mutable {
                auto idx = reinterpret_bits<int>(verts(planeConsIDTag,vi));
                if(idx < 0)
                    return;      
                auto ktri = ktris.pack(dim_c<3>,"inds",idx).reinterpret_bits(int_c);

                auto is_inverted_vert = vtemp("is_inverted",vi) > (T)0.5;
                if(is_inverted_vert)
                    return;



                auto plane_root = kverts.pack(dim_c<3>,"x",ktri[0]);
                auto plane_nrm = ktris.pack(dim_c<3>,"nrm",idx);

                auto mu = verts("mu",vi);
                auto lam = verts("lam",vi);
                    // if(distance > collisionEps)
                auto eps = kine_out_collisionEps;
                auto p = vtemp.pack(dim_c<3>,"xn",vi);
                auto seg = p - plane_root;

                auto fc = vec3::zeros();
                auto Hc = mat3::zeros();
                auto dist = seg.dot(plane_nrm) - eps;
                if(dist < (T)0 || use_sticky_condition){
                    fc = -dist * mu * plane_constraint_stiffness * plane_nrm;
                    Hc = mu * plane_constraint_stiffness * dyadic_prod(plane_nrm,plane_nrm);
                }

                // printf("apply plane constraint with force : %f %f\n",(float)dist,(float)fc.norm());

                nodal_gh_buffer.tuple(dim_c<3>,"grad",vi) = fc;
                nodal_gh_buffer.tuple(dim_c<3,3>,"H",vi) = Hc;

            });


            cudaPol(zs::range(tris.size()),[
                    vtemp = proxy<space>({},vtemp),
                    sttemp = proxy<space>({},sttemp),
                    verts = proxy<space>({},verts),
                    tris = proxy<space>({},tris),
                    kverts = proxy<space>({},kverts),
                    ktris = proxy<space>({},ktris),
                    cnorm = cnorm,
                    planeConsIDTag = zs::SmallString(planeConsIDTag),
                    kine_out_collisionEps = kine_out_collisionEps,
                    kine_in_collisionEps = kine_in_collisionEps,
                    plane_constraint_stiffness = plane_constraint_stiffness,
                    use_sticky_condition = use_sticky_condition,
                    tris_gh_buffer = proxy<space>({},tris_gh_buffer)] ZS_LAMBDA(int ti) mutable {
                auto kp_idx = reinterpret_bits<int>(tris(planeConsIDTag,ti));
                if(kp_idx < 0)
                    return;
                auto kp = kverts.pack(dim_c<3>,"x",kp_idx);
                auto tri = tris.pack(dim_c<3>,"inds",ti).reinterpret_bits(int_c);
                for(int i = 0;i != 3;++i){
                    auto is_inverted_vert = vtemp("is_inverted",tri[i]) > (T)0.5;
                    if(is_inverted_vert)
                        return;
                }
            
                // auto tnrm = sttemp.pack(dim_c<3>,"nrm",ti);

                auto mu = verts("mu",tri[0]);
                auto lam = verts("lam",tri[0]);

                auto eps = kine_out_collisionEps;
                vec3 vs[4] = {};
                vs[0] = kp;
                for(int i = 0;i != 3;++i)
                    vs[i + 1] = vtemp.pack(dim_c<3>,"xn",tri[i]);
                
                vec3 e[3] = {};
                e[0] = vs[3] - vs[2];
                e[1] = vs[0] - vs[2];
                e[2] = vs[1] - vs[2];

                auto n = e[2].cross(e[0]);
                // if(n.norm() < 1e-4)
                //     return;
                n = n/(n.norm() + 1e-6);

                T springLength = e[1].dot(n) - eps;
                auto gvf = zs::vec<T,9>::zeros();
                if(springLength < (T)0 || use_sticky_condition){
                    auto gvf_v12 = COLLISION_UTILS::springLengthGradient(vs,e,n);
                    if(isnan(gvf_v12.norm()))
                        printf("nan gvf detected at %d %f %f\n",ti,gvf_v12.norm(),n.norm());
                    for(int i = 0;i != 9;++i)
                        gvf[i] = gvf_v12[i + 3];
                }
                cnorm = (T)1.0;
                auto stiffness = plane_constraint_stiffness * cnorm;
                // stiffness = (T)0;            
                auto g = -stiffness * (T)2.0 * mu * springLength * gvf;
                auto H = stiffness * (T)2.0 * mu * zs::dyadic_prod(gvf, gvf);
                
                // if(springLength < (T)0) {
                //     auto springLengthH_M12 = COLLISION_UTILS::springLengthHessian(vs,e,n);
                //     auto springLengthH_M9 = mat9::zeros();
                //     for(int r = 0;r != 9;++r)
                //         for(int c = 0;c != 9;++c)
                //             springLengthH_M9(r,c) = springLengthH_M12(r + 3,c+ 3);
                //     H += springLength * springLengthH_M9 * (T)2.0 * stiffness * mu;
                //     make_pd(H);
                // }

                tris_gh_buffer.tuple(dim_c<9>,"grad",ti) = g;
                tris_gh_buffer.tuple(dim_c<9,9>,"H",ti) = H;           
            });
        }

        void  computePlaneConstraintGradientAndHessian(zs::CudaExecutionPolicy& cudaPol,
                            const dtiles_t& vtemp,
                            const std::string& planeConsPosTag,
                            const std::string& planeConsNrmTag,
                            const std::string& planeConsIDTag,
                            dtiles_t& nodal_gh_buffer) {
            using namespace zs;
            constexpr auto space = execspace_e::cuda;

            cudaPol(zs::range(verts.size()),[
                    verts = proxy<space>({},verts),
                    vtemp = proxy<space>({},vtemp),
                    planeConsPosTag = zs::SmallString(planeConsPosTag),
                    planeConsNrmTag = zs::SmallString(planeConsNrmTag),
                    planeConsIDTag = zs::SmallString(planeConsIDTag),
                    kine_out_collisionEps = kine_out_collisionEps,
                    plane_constraint_stiffness = plane_constraint_stiffness,
                    nodal_gh_buffer = proxy<space>({},nodal_gh_buffer)] ZS_LAMBDA(int vi) mutable {
                auto idx = reinterpret_bits<int>(verts(planeConsIDTag,vi));
                if(idx < 0)
                    return;

                // if(kverts.hasProperty("k_fail"))
                // if(verts("is_inverted",vi) > (T)0.5)
                //     return;


                auto mu = verts("mu",vi);
                auto lam = verts("lam",vi);

                auto eps = kine_out_collisionEps;
                auto plane_nrm = verts.pack(dim_c<3>,planeConsNrmTag,vi);
                auto plane_root = verts.pack(dim_c<3>,planeConsPosTag,vi);

                auto p = vtemp.pack(dim_c<3>,"xn",vi);
                auto seg = p - plane_root;

                auto fc = vec3::zeros();
                auto Hc = mat3::zeros();
                auto dist = seg.dot(plane_nrm) - eps;
                if(dist < (T)0){
                    fc = -dist * mu * plane_constraint_stiffness * plane_nrm;
                    Hc = mu * plane_constraint_stiffness * dyadic_prod(plane_nrm,plane_nrm);
                }

                // printf("apply plane constraint with force : %f %f\n",(float)dist,(float)fc.norm());

                nodal_gh_buffer.tuple(dim_c<3>,"grad",vi) = fc;
                nodal_gh_buffer.tuple(dim_c<3,3>,"H",vi) = Hc;
            });

            // cudaPol(zs::range(tris.size()),[
            //         verts = proxy<space>({},verts),
            //         tris = proxy<space>({},tris),
            //         vtemp = proxy<space>({},vtemp),
            //         planeConsPosTag = zs::SmallString(planeConsPosTag),
            //         planeConsNrmTag = zs::SmallString(planeConsNrmTag),
            //         planeConsIDTag = zs::SmallString(planeConsIDTag),
            //         kine_out_collisionEps = kine_out_collisionEps,
            //         plane_constraint_stiffness = plane_constraint_stiffness,
            //         nodal_gh_buffer = proxy<space>({},nodal_gh_buffer)] ZS_LAMBDA(int ti) mutable {
            //     auto idx = reinterpret_bits<int>(tris(planeConsIDTag,ti));
            //     if(idx < 0)
            //         return;

            //     auto tri = tris.pack(dim_c<3>,"inds",ti).reinterpret_bits(int_c);

            //     auto mu = verts("mu",tri[0]);
            //     auto lam = verts("lam",tri[0]);

            //     auto eps = kine_out_collisionEps * 2.0;
            //     auto plane_nrm = tris.pack(dim_c<3>,planeConsNrmTag,ti);
            //     auto plane_root = tris.pack(dim_c<3>,planeConsPosTag,ti);

            //     auto p = vec3::zeros();
            //     for(int i = 0;i != 3;++i)
            //         p += vtemp.pack(dim_c<3>,"xn",tri[i])/(T)3.0;
            //     auto seg = p - plane_root;

            //     auto fc = vec3::zeros();
            //     auto Hc = mat3::zeros();
            //     auto dist = seg.dot(plane_nrm) - eps;
            //     if(dist < (T)0){
            //         fc = -dist * mu * plane_constraint_stiffness * plane_nrm;
            //         Hc = mu * plane_constraint_stiffness * dyadic_prod(plane_nrm,plane_nrm);
            //     }

            //     // printf("apply plane constraint with force : %f %f\n",(float)dist,(float)fc.norm());
            //     for(int i = 0;i != 3;++i) {
            //         auto vi = tri[i];
            //         for(int d = 0;d != 3;++d)
            //             atomic_add(exec_cuda,&nodal_gh_buffer("grad",d,vi),fc[d]/(T)3.0);
            //         for(int r = 0;r != 3;++r)
            //             for(int c = 0;c != 3;++c)
            //                 atomic_add(exec_cuda,&nodal_gh_buffer("H",r * 3 + c,vi),Hc(r,c)/(T)9.0);
            //     }

            //     // nodal_gh_buffer.tuple(dim_c<3>,"grad",vi) = fc;
            //     // nodal_gh_buffer.tuple(dim_c<3,3>,"H",vi) = Hc;
            // });
        }

        template <typename ElasticModel,typename AnisoElasticModel>
        void computeGradientAndHessian(zs::CudaExecutionPolicy& cudaPol,
                            const ElasticModel& model,
                            const AnisoElasticModel& amodel,
                            const dtiles_t& vtemp,
                            const dtiles_t& etemp,
                            dtiles_t& gh_buffer,
                            T kd_alpha = (T)0.0,
                            T kd_beta = (T)0.0) {        
            using namespace zs;
            constexpr auto space = execspace_e::cuda;

            int offset = 0;
            TILEVEC_OPS::copy<4>(cudaPol,eles,"inds",gh_buffer,"inds",offset);   
            // eval the inertia term gradient
            // cudaPol(zs::range(eles.size()),[dt2 = dt2,
            //             verts = proxy<space>({},verts),
            //             eles = proxy<space>({},eles),
            //             vtemp = proxy<space>({},vtemp),
            //             gh_buffer = proxy<space>({},gh_buffer),
            //             dt = dt,offset = offset] ZS_LAMBDA(int ei) mutable {
            //     auto m = eles("m",ei)/(T)4.0;
            //     auto inds = eles.pack(dim_c<4>,"inds",ei).reinterpret_bits(int_c);
            //     auto pgrad = zs::vec<T,12>::zeros();
            //     // auto H  = zs::vec<T,12,12>::zeros();
            //     // if(eles.hasProperty("dt")) {
            //     //     dt2 = eles("dt",ei) * eles("dt",ei);
            //     // }

            //     auto inertia = (T)1.0;
            //     if(eles.hasProperty("inertia"))
            //         inertia = eles("inertia",ei);
            //     for(int i = 0;i != 4;++i){
            //         auto x1 = vtemp.pack(dim_c<3>,"xn",inds[i]);
            //         auto x0 = vtemp.pack(dim_c<3>,"xp",inds[i]);
            //         auto v0 = vtemp.pack(dim_c<3>,"vp",inds[i]);

            //         auto alpha = inertia * m/dt2;
            //         auto nodal_pgrad = -alpha * (x1 - x0 - v0 * dt);
            //         for(int d = 0;d != 3;++d){
            //             auto idx = i * 3 + d;
            //             gh_buffer("grad",idx,ei) = nodal_pgrad[d];
            //             gh_buffer("H",idx*12 + idx,ei + offset) = alpha;
            //         }
                    
            //     }
            //     // gh_buffer.tuple(dim_c<12>,"grad",ei + offset) = pgrad;
            //     // gh_buffer.template tuple<12*12>("H",ei + offset) = H;
            // });


            cudaPol(zs::range(eles.size()), [dt = dt,dt2 = dt2,aniso_strength = aniso_strength,
                            verts = proxy<space>({},verts),
                            vtemp = proxy<space>({}, vtemp),
                            etemp = proxy<space>({}, etemp),
                            gh_buffer = proxy<space>({},gh_buffer),
                            eles = proxy<space>({}, eles),
                            kd_alpha = kd_alpha,kd_beta = kd_beta,
                            model = model,amodel = amodel, volf = volf,offset = offset] ZS_LAMBDA (int ei) mutable {
                auto DmInv = eles.pack(dim_c<3,3>,"IB",ei);
                auto dFdX = dFdXMatrix(DmInv);
                auto inds = eles.pack(dim_c<4>,"inds",ei).reinterpret_bits(int_c);
                vec3 x1[4] = {vtemp.pack(dim_c<3>,"xn", inds[0]),
                                vtemp.pack(dim_c<3>,"xn", inds[1]),
                                vtemp.pack(dim_c<3>,"xn", inds[2]),
                                vtemp.pack(dim_c<3>,"xn", inds[3])};


                mat3 FAct{};
                mat3 F{};
                {
                    auto x1x0 = x1[1] - x1[0];
                    auto x2x0 = x1[2] - x1[0];
                    auto x3x0 = x1[3] - x1[0];
                    auto Ds = mat3{x1x0[0], x2x0[0], x3x0[0], x1x0[1], x2x0[1],
                                    x3x0[1], x1x0[2], x2x0[2], x3x0[2]};
                    F = Ds * DmInv;
                    FAct = F * etemp.template pack<3,3>("ActInv",ei);
                } 
                auto dFActdF = dFAdF(etemp.template pack<3,3>("ActInv",ei));

                // add the force term in gradient
                if(eles.hasProperty("mu") && eles.hasProperty("lam")) {
                    model.mu = eles("mu",ei);
                    model.lam = eles("lam",ei);
                }

                auto inversion_strength = (T)1.0;
                // for(int i = 0;i != 4;++i)
                //     if(vtemp("is_inverted",inds[i]) < (T)0.5)
                //         inversion_strength = (T)1.0;

                auto P = model.first_piola(FAct) * inversion_strength;
                auto vole = eles("vol", ei);
                auto vecP = flatten(P);
                vecP = dFActdF.transpose() * vecP;
                auto dFdXT = dFdX.transpose();
                auto vf = -vole * (dFdXT * vecP);     

                auto mg = volf * vole / (T)4.0;
                for(int i = 0;i != 4;++i)
                    for(int d = 0;d !=3 ;++d){
                        vf[i*3 + d] += mg[d];
                    }


                // assemble element-wise hessian matrix
                auto Hq = model.first_piola_derivative(FAct, true_c) * inversion_strength;
                auto dFdAct_dFdX = dFActdF * dFdX; 
                // add inertia hessian term
                auto H = dFdAct_dFdX.transpose() * Hq * dFdAct_dFdX * vole;

                if(isnan(H.norm())) {
                    printf("nan CH detected at Elastic : %d %f %f %f %f\nFAct = \n%f %f %f\n%f %f %f\n%f %f %f\nF = \n%f %f %f\n%f %f %f\n%f %f %f\n",ei,
                        (float)Hq.norm(),
                        (float)dFdAct_dFdX.norm(),
                        (float)P.norm(),
                        (float)FAct.norm(),
                        (float)FAct(0,0),(float)FAct(0,1),(float)FAct(0,2),
                        (float)FAct(1,0),(float)FAct(1,1),(float)FAct(1,2),
                        (float)FAct(2,0),(float)FAct(2,1),(float)FAct(2,2),
                        (float)F(0,0),(float)F(0,1),(float)F(0,2),
                        (float)F(1,0),(float)F(1,1),(float)F(1,2),
                        (float)F(2,0),(float)F(2,1),(float)F(2,2)                     
                    );
                }



                // if(eles.hasProperty("Muscle_ID") && (int)eles("Muscle_ID",ei) >= 0) {
                //     auto fiber = eles.pack(dim_c<3>,"fiber",ei);
                //     if(zs::abs(fiber.norm() - 1.0) < 1e-3) {
                //         fiber /= fiber.norm();
                //         // if(eles.hasProperty("mu")) {
                //         //     amodel.mu = eles("mu",ei);
                //         //     // amodel.lam = eles("lam",ei);
                            
                //         // }
                //         auto aP = amodel.do_first_piola(FAct,fiber);
                //         auto vecAP = flatten(P);
                //         vecAP = dFActdF.transpose() * vecP;
                //         vf -= vole  * dFdXT * vecAP *aniso_strength;

                //         auto aHq = amodel.do_first_piola_derivative(FAct,fiber);
                //         H += dFdAct_dFdX.transpose() * aHq * dFdAct_dFdX * vole * aniso_strength;
                //         // if((int)eles("Muscle_ID",ei) == 0){
                //         //     printf("fiber : %f %f %f,Fa = %f,aP = %f,aHq = %f,H = %f\n",fiber[0],fiber[1],fiber[2],(float)FAct.norm(),(float)aP.norm(),(float)aHq.norm(),(float)H.norm());
                //         // }
                //     }
                // }


                // adding rayleigh damping term
                // vec3 v0[4] = {vtemp.pack(dim_c<3>,"vn", inds[0]),
                // vtemp.pack(dim_c<3>,"vn", inds[1]),
                // vtemp.pack(dim_c<3>,"vn", inds[2]),
                // vtemp.pack(dim_c<3>,"vn", inds[3])}; 

                // auto inertia = (T)1.0;
                // if(eles.hasProperty("inertia"))
                //     inertia = eles("inertia",ei);

                // auto vel = COLLISION_UTILS::flatten(v0); 
                // auto m = eles("m",ei)/(T)4.0;
                // auto C = kd_beta * H + kd_alpha * inertia * m * zs::vec<T,12,12>::identity();
                // auto rdamping = C * vel;  

                gh_buffer.tuple(dim_c<12>,"grad",ei + offset) = gh_buffer.pack(dim_c<12>,"grad",ei + offset) + vf/* - rdamping*/; 
                // gh_buffer.tuple(dim_c<12>,"grad",ei + offset) = gh_buffer.pack(dim_c<12>,"grad",ei + offset) - rdamping; 
                // H += kd_beta*H/dt;

                gh_buffer.template tuple<12*12>("H",ei + offset) = gh_buffer.template pack<12,12>("H",ei + offset) + H/* + C/dt*/;
            });
        // Bone Driven Potential Energy
            // T lambda = model.lam;
            // T mu = model.mu;

            auto nmEmbedVerts = b_verts.size();

            // TILEVEC_OPS::fill_range<4>(cudaPol,gh_buffer,"inds",zs::vec<int,4>::uniform(-1).reinterpret_bits(float_c),eles.size() + offset,b_verts.size());
            // TILEVEC_OPS::fill_range<3>(cudaPol,gh_buffer,"grad",zs::vec<T,3>::zeros(),eles.size() + offset,b_verts.size());
            // TILEVEC_OPS::fill_range<144>(cudaPol,gh_buffer,"H",zs::vec<T,144>::zeros(),eles.size() + offset,b_verts.size());

            // we should neglect the inverted element
            // std::cout << "nmEmbedVerts : " << nmEmbedVerts << std::endl;
            // std::cout << "bcwsize :  " << b_bcws.size() << std::endl;
            // return;
            cudaPol(zs::range(nmEmbedVerts), [
                    gh_buffer = proxy<space>({},gh_buffer),model = model,
                    bcws = proxy<space>({},b_bcws),b_verts = proxy<space>({},b_verts),vtemp = proxy<space>({},vtemp),etemp = proxy<space>({},etemp),
                    eles = proxy<space>({},eles),bone_driven_weight = bone_driven_weight,offset = offset] ZS_LAMBDA(int vi) mutable {
                        auto ei = reinterpret_bits<int>(bcws("inds",vi));
 
                        if(ei < 0){

                            return;
                        }
                        // if(ei >= etemp.size()){
                        //     printf("ei too big for etemp\n");
                        //     return;
                        // }
                        // auto is_inverted = reinterpret_bits<int>(etemp("is_inverted",ei));
                        // if(is_inverted){
                        //     if(vi == 0)
                        //         printf("inverted tet\n");
                        //     return;
                        // }

                        // auto FatID = eles("FatID",ei);
                        // if(FatID > 0)
                        //     return;

                        auto lambda = model.lam;
                        auto mu = model.mu;
                        // if(eles.hasProperty("mu") && eles.hasProperty("lam")) {
                        //     mu = eles("mu",ei);
                        //     lambda = eles("lam",ei);
                        // }

                        auto inds = eles.pack(dim_c<4>,"inds",ei).reinterpret_bits(int_c);
                        // gh_buffer.tuple(dim_c<4>,"inds",vi + offset + eles.size()) = eles.pack(dim_c<4>,"inds",ei);
                        auto w = bcws.pack(dim_c<4>,"w",vi);
                        // if(w[0] < 1e-4 || w[1] < 1e-4 || w[2] < 1e-4 || w[3] < 1e-4){
                        //     // if(vi == 0)
                        //     //     printf("boundary tet\n");
                        //     return;
                        // }
                        auto tpos = vec3::zeros();
                        for(int i = 0;i != 4;++i)
                            tpos += w[i] * vtemp.pack(dim_c<3>,"xn",inds[i]);
                        auto pdiff = tpos - b_verts.pack<3>("x",vi);
                        // auto pdiff = tpos - b_verts[vi];

                        T stiffness = (2.0066 * mu + 1.0122 * lambda) * b_verts("strength",vi);

                        // zs::vec<T,12> elm_grad{};
                        // auto elm_H = zs::vec<T,12,12>::zeros();

                        // if(vi == 0) {
                        //     printf("stiff : %f dw : %f strength : %f cnorm : %f vol : %f bdw : %f\n",
                        //         (float)stiffness,
                        //         (float)bone_driven_weight,
                        //         (float)bcws("strength",vi),
                        //         (float)bcws("cnorm",vi),
                        //         (float)eles("vol",ei),
                        //         (float)eles("bdw",ei));
                        // }

                        auto alpha = stiffness * bone_driven_weight * bcws("strength",vi) * bcws("cnorm",vi) * eles("vol",ei) * eles("bdw",ei);

                        for(size_t i = 0;i != 4;++i){
                            auto tmp = -pdiff * alpha * w[i]; 
                            // if(vi == 0 && i == 0) {
                                // printf("check: %f %f %f\n",(float)tmp[0],(float)tmp[1],(float)tmp[2]);
                            // }
                            for(size_t d = 0;d != 3;++d){
                                atomic_add(exec_cuda,&gh_buffer("grad",i*3 + d,ei),tmp[d]);
                                // elm_grad[i*3 + d] = tmp[d];
                                // atomic_add(exec_cuda,&gh_buffer("grad",i * 3 + d,ei),tmp[d]);
                            }
                        }
                        for(int i = 0;i != 4;++i)
                            for(int j = 0;j != 4;++j){
                                T beta = alpha * w[i] * w[j];
                                if(isnan(beta))
                                    printf("nan H detected at driver : %d\n",vi);
                                for(int d = 0;d != 3;++d){
                                    atomic_add(exec_cuda,&gh_buffer("H",(i*3 + d)*12 + j*3 + d,ei),beta);
                                }
                            }
                        
                        // for(int i = 0;i != 12;++i){
                            // atomic_add(exec_cuda,&gh_buffer("grad",i,ei),elm_grad[i]);
                            // for(int j = 0;j != 12;++j)
                            //     atomic_add(exec_cuda,&gh_buffer("H",i*12 + j,ei),elm_H(i,j));
                        // }
                        // gh_buffer.tuple(dim_c<12>,"grad",vi + eles.size() + offset) = elm_grad;
                        // gh_buffer.tuple(dim_c<12*12>,"H",vi + eles.size() + offset) = elm_H;
            });

            // cudaPol(zs::range(eles.size()), [gh_buffer = proxy<space>({},gh_buffer)] ZS_LAMBDA (int ei) mutable {
            //     auto H = gh_buffer.template pack<12,12>("H",ei);
            //     make_pd(H);
            //     gh_buffer.template tuple<12*12>("H",ei) = H;
            // });

        }

        template <typename ElasticModel>
        void computeElasticBonesEnergy(zs::CudaExecutionPolicy& cudaPol,
                            const ElasticModel& model,    
                            const dtiles_t& vtemp,
                            const dtiles_t& etemp,
                            T& res) {
            using namespace zs;
            constexpr auto space = execspace_e::cuda;

            Vector<T> psi{vtemp.get_allocator(), 1};
            psi.setVal((T)0);
            cudaPol(zs::range(eles.size()), [
                            verts = proxy<space>({},verts),
                            vtemp = proxy<space>({}, vtemp),
                            etemp = proxy<space>({}, etemp),
                            psi = proxy<space>(psi),
                            eles = proxy<space>({}, eles),
                            model = model] ZS_LAMBDA (int ei) mutable {
                auto DmInv = eles.pack(dim_c<3,3>,"IB",ei);
                auto dFdX = dFdXMatrix(DmInv);
                auto inds = eles.pack(dim_c<4>,"inds",ei).reinterpret_bits(int_c);
                vec3 x1[4] = {vtemp.pack(dim_c<3>,"xn", inds[0]),
                                vtemp.pack(dim_c<3>,"xn", inds[1]),
                                vtemp.pack(dim_c<3>,"xn", inds[2]),
                                vtemp.pack(dim_c<3>,"xn", inds[3])};   
                mat3 FAct{};
                {
                    auto x1x0 = x1[1] - x1[0];
                    auto x2x0 = x1[2] - x1[0];
                    auto x3x0 = x1[3] - x1[0];
                    auto Ds = mat3{x1x0[0], x2x0[0], x3x0[0], x1x0[1], x2x0[1],
                                    x3x0[1], x1x0[2], x2x0[2], x3x0[2]};
                    FAct = Ds * DmInv;
                    FAct = FAct * etemp.template pack<3,3>("ActInv",ei);
                } 
                auto inversion_strength = (T)1.0;
                // for(int i = 0;i != 4;++i)
                //     if(vtemp("is_inverted",inds[i]) < (T)0.5)
                //         inversion_strength = (T)1.0;
                auto vole = eles("vol", ei);
                auto epsi = vole * model.psi(FAct) * inversion_strength;                

                atomic_add(exec_cuda,&psi[0],epsi);
            });

            auto nmEmbedVerts = b_verts.size();
            cudaPol(zs::range(nmEmbedVerts), [
                    model = model,psi = proxy<space>(psi),
                    bcws = proxy<space>({},b_bcws),b_verts = proxy<space>({},b_verts),vtemp = proxy<space>({},vtemp),etemp = proxy<space>({},etemp),
                    eles = proxy<space>({},eles),bone_driven_weight = bone_driven_weight] ZS_LAMBDA(int vi) mutable {
                        auto ei = reinterpret_bits<int>(bcws("inds",vi));
 
                        if(ei < 0){
                            return;
                        }

                        auto lambda = model.lam;
                        auto mu = model.mu;

                        auto inds = eles.pack(dim_c<4>,"inds",ei).reinterpret_bits(int_c);
                        // gh_buffer.tuple(dim_c<4>,"inds",vi + offset + eles.size()) = eles.pack(dim_c<4>,"inds",ei);
                        auto w = bcws.pack(dim_c<4>,"w",vi);

                        auto tpos = vec3::zeros();
                        for(int i = 0;i != 4;++i)
                            tpos += w[i] * vtemp.pack(dim_c<3>,"xn",inds[i]);
                        auto pdiff = tpos - b_verts.pack<3>("x",vi);
                        // auto pdiff = tpos - b_verts[vi];

                        T stiffness = (2.0066 * mu + 1.0122 * lambda) * b_verts("strength",vi);

                        auto alpha = stiffness * bone_driven_weight * bcws("strength",vi) * bcws("cnorm",vi) * eles("vol",ei) * eles("bdw",ei);
                        T bpsi = (T)0.5 * pdiff.l2NormSqr() * alpha; 

                        atomic_add(exec_cuda,&psi[0],bpsi);

            });            


            res = psi.getVal();
        }

        FEMDynamicSteppingSystem(const tiles_t &verts, const tiles_t &eles,
                const tiles_t& points,const tiles_t& lines,const tiles_t& tris,
                T in_collisionEps,T out_collisionEps,
                const tiles_t &b_bcws, const tiles_t& b_verts,T bone_driven_weight,
                const vec3& volf,const T& _dt,const T& collisionStiffness,
                const T& kine_in_collisionEps,const T& kine_out_collisionEps,
                const T& kineCollisionStiffness,const T& aniso_strength,const T& binderStiffness,const T& plane_constraint_stiffness)
            : verts{verts}, eles{eles},points{points}, lines{lines}, tris{tris},
                    in_collisionEps{in_collisionEps},out_collisionEps{out_collisionEps},
                    b_bcws{b_bcws}, b_verts{b_verts}, bone_driven_weight{bone_driven_weight},
                    volf{volf},binderStiffness{binderStiffness},plane_constraint_stiffness{plane_constraint_stiffness},
                    kine_in_collisionEps{kine_in_collisionEps},kine_out_collisionEps{kine_out_collisionEps},
                    kineCollisionStiffness{kineCollisionStiffness},aniso_strength{aniso_strength},
                    dt{_dt}, dt2{_dt * _dt},collisionStiffness{collisionStiffness},use_edge_edge_collision{true}, use_vertex_facet_collision{true} {}

        const tiles_t &verts;
        const tiles_t &eles;
        const tiles_t &points;
        const tiles_t &lines;
        const tiles_t &tris;
        const tiles_t &b_bcws;  // the barycentric interpolation of embeded bones 
        const tiles_t &b_verts; // the position of embeded bones

        T bone_driven_weight;
        vec3 volf;
        T dt;
        T dt2;
        T in_collisionEps;
        T out_collisionEps;

        T collisionStiffness;

        bool bvh_initialized;
        bool use_edge_edge_collision;
        bool use_vertex_facet_collision;

        T kine_in_collisionEps;
        T kine_out_collisionEps;
        T kineCollisionStiffness;

        T aniso_strength;


        T binderStiffness;
        // int default_muscle_id;
        // zs::vec<T,3> default_muscle_dir;
        // T default_act;

        // T inset;
        // T outset;

        T plane_constraint_stiffness;
    };




    void apply() override {
        using namespace zs;
        auto zsparticles = get_input<ZenoParticles>("ZSParticles");
        auto gravity = zeno::vec<3,T>(0);
        if(has_input("gravity"))
            gravity = get_input2<zeno::vec<3,T>>("gravity");
        // T armijo = (T)1e-4;
        // T wolfe = (T)0.9;
        // T cg_res = (T)0.01;
        // T cg_res = (T)0.0001;
        // T cg_res = get_param<float>("cg_res");
        T cg_res = get_input2<float>("cg_res");
        T btl_res = (T)0.1;
        auto models = zsparticles->getModel();
        auto& verts = zsparticles->getParticles();
        auto& eles = zsparticles->getQuadraturePoints();

        // zs::Vector<vec3>(MAX_VERTS)
        // TileVec("pos","tag","deleted","")

        if(eles.getPropertySize("inds") != 4)
            throw std::runtime_error("the input zsparticles is not a tetrahedra mesh");
        if(!zsparticles->hasAuxData(ZenoParticles::s_surfTriTag))
            throw std::runtime_error("the input zsparticles has no surface tris");
        if(!zsparticles->hasAuxData(ZenoParticles::s_surfEdgeTag))
            throw std::runtime_error("the input zsparticles has no surface lines");
        if(!zsparticles->hasAuxData(ZenoParticles::s_surfVertTag)) 
            throw std::runtime_error("the input zsparticles has no surface points");
        if(!zsparticles->hasAuxData(ZenoParticles::s_surfHalfEdgeTag))
            throw std::runtime_error("the input zsparticles has no half edge structures");

        auto& tris  = (*zsparticles)[ZenoParticles::s_surfTriTag];
        auto& lines = (*zsparticles)[ZenoParticles::s_surfEdgeTag];
        auto& points = (*zsparticles)[ZenoParticles::s_surfVertTag];
        const auto& halfedges = (*zsparticles)[ZenoParticles::s_surfHalfEdgeTag];

        auto muscle_id_tag = get_input2<std::string>("muscle_id_tag");

        // auto bone_driven_weight = (T)0.02;

        auto newton_res = get_input2<float>("newton_res");

        auto dt = get_input2<float>("dt");

        auto volf = vec3::from_array(gravity * models.density);

        std::vector<zeno::vec2f> act_;    
        std::size_t nm_acts = 0;

        if(has_input("Acts")) {
            act_ = get_input<zeno::ListObject>("Acts")->getLiterial<zeno::vec2f>();
            nm_acts = act_.size();
        }

        std::cout << "nmActs:" << nm_acts << std::endl;

        constexpr auto host_space = zs::execspace_e::openmp;
        auto ompExec = zs::omp_exec();
        auto act_buffer = dtiles_t{{{"act",2}},nm_acts,zs::memsrc_e::host};
        ompExec(zs::range(act_buffer.size()),
            [act_buffer = proxy<host_space>({},act_buffer),act_] (int i) mutable {
                act_buffer.tuple(dim_c<2>,"act",i) = vec2(act_[i][0],act_[i][1]);
        });

        act_buffer = act_buffer.clone({zs::memsrc_e::device, 0});

        auto driven_tag = get_input2<std::string>("driven_tag");
        auto bone_driven_weight = get_input2<float>("driven_weight");


        constexpr auto space = execspace_e::cuda;
        auto cudaPol = cuda_exec();

        auto bbw = typename ZenoParticles::particles_t({
            {"X",3},
            {"inds",1},
            {"w",4},
            {"strength",1},
            {"cnorm",1}},0,zs::memsrc_e::device,0);

        auto bverts = typename ZenoParticles::particles_t({
            {"x",3},
            {"strength",1}},0,zs::memsrc_e::device,0);
        if(has_input<ZenoParticles>("driven_boudary") && zsparticles->hasAuxData(driven_tag)){
            auto zsbones = get_input<ZenoParticles>("driven_boudary");
            const auto& zsbones_verts = zsbones->getParticles();
            bverts.resize(zsbones_verts.size());


            TILEVEC_OPS::copy(cudaPol,zsbones_verts,"x",bverts,"x");
            if(zsbones_verts.hasProperty("strength"))
                TILEVEC_OPS::copy(cudaPol,zsbones_verts,"strength",bverts,"strength");
            else   
                TILEVEC_OPS::fill(cudaPol,bverts,"strength",(T)1.0);

            const auto& inbbw = (*zsparticles)[driven_tag];
            bbw.resize(inbbw.size());
            TILEVEC_OPS::copy(cudaPol,inbbw,"X",bbw,"X");
            TILEVEC_OPS::copy(cudaPol,inbbw,"inds",bbw,"inds");
            TILEVEC_OPS::copy(cudaPol,inbbw,"w",bbw,"w");
            TILEVEC_OPS::copy(cudaPol,inbbw,"strength",bbw,"strength");
            TILEVEC_OPS::copy(cudaPol,inbbw,"cnorm",bbw,"cnorm");

            // if(zsbones_verts.has_attr<float>("drivenStrength"))
            //     ompExec(zs::range(zsbones_verts.size()),
            //         [bverts = proxy<host_space>(bverts),&zsbones_verts] (int i) mutable {
            //             auto v = zsbones_verts[i];
            //             bverts[i] = zs::vec<T,3>{v[0],v[1],v[2]};
            //     });

        }
        // bverts = bverts.clone({zs::memsrc_e::device,0});
        // std::cout << "bverts.size() = " << bverts.size() << std::endl;

        auto kverts = typename ZenoParticles::particles_t({
                {"x",3},
                {"xp",3},
                {"b_fail",1},
                {"binderStiffness",1},
                {"nrm",3},
                {"area",1}},0,zs::memsrc_e::device,0);
        auto ktris = typename ZenoParticles::particles_t({
                {"inds",3},
                {"nrm",3}},0,zs::memsrc_e::device,0);


        dtiles_t surf_tris_buffer{tris.get_allocator(),{
            {"inds",3},
            {"nrm",3},
            {"he_inds",1}
        },tris.size()};

        dtiles_t surf_verts_buffer{points.get_allocator(),{
            {"inds",1},
            {"xn",3}
        },points.size()};
        TILEVEC_OPS::copy(cudaPol,points,"inds",surf_verts_buffer,"inds");
        TILEVEC_OPS::copy(cudaPol,tris,"inds",surf_tris_buffer,"inds");
        TILEVEC_OPS::copy(cudaPol,tris,"he_inds",surf_tris_buffer,"he_inds");
        reorder_topology(cudaPol,points,surf_tris_buffer);
        // zs::Vector<int> nodal_colors{surf_verts_buffer.get_allocator(),surf_verts_buffer.size()};
        dtiles_t gia_res{points.get_allocator(),{
            {"ring_mask",1},
            {"type_mask",1},
            {"color_mask",1}
        },points.size()};
        // zs::Vector<zs::vec<int,2>> instBuffer{surf_verts_buffer.get_allocator(),surf_verts_buffer.size() * 8};
        dtiles_t inst_buffer_info{tris.get_allocator(),{
            {"pair",2},
            {"type",1},
            {"its_edge_mark",6},
            {"int_points",6}
        },tris.size() * 2};


        if(has_input<ZenoParticles>("kinematic_boundary")){
            auto kinematic_boundary = get_input<ZenoParticles>("kinematic_boundary");
            // if (kinematic_boundary.empty())

            // const auto& prim_kverts = kinematic_boundary.verts;
            // auto& prim_kverts_area = kinematic_boundary.attr<float>("area");
            auto& kb_verts = kinematic_boundary->getParticles();
            auto& kb_tris = kinematic_boundary->getQuadraturePoints();

            // auto& kb_tris = kinematic_boundary->getQuadraturePoints();
            // if(kb_tris.getPropertySize("inds") != 3){
            //     fmt::print(fg(fmt::color::red),"the kinematic boundary is not a surface triangulate mesh\n");
            //     throw std::runtime_error("the kinematic boundary is not a surface triangulate mesh");
            // }
            // if(!kb_tris.hasProperty("area")){
            //     fmt::print(fg(fmt::color::red),"the kinematic boundary has no 'area' channel\n");
            //     throw std::runtime_error("the kinematic boundary has no 'area' channel");
            // }     
            kverts.resize(kb_verts.size());
            TILEVEC_OPS::copy<3>(cudaPol,kb_verts,"x",kverts,"x");
            TILEVEC_OPS::copy<3>(cudaPol,kb_verts,"x",kverts,"xp");
            TILEVEC_OPS::copy<3>(cudaPol,kb_verts,"nrm",kverts,"nrm");
            TILEVEC_OPS::fill(cudaPol,kverts,"area",(T)1.0);
            if(kb_verts.hasProperty("b_fail"))
                TILEVEC_OPS::copy(cudaPol,kb_verts,"b_fail",kverts,"b_fail");
            else 
                TILEVEC_OPS::fill(cudaPol,kverts,"b_fail",(T)0.0);
            if(kb_verts.hasProperty("binderStiffness"))
                TILEVEC_OPS::copy(cudaPol,kb_verts,"binderStiffness",kverts,"binderStiffness");
            else 
                TILEVEC_OPS::fill(cudaPol,kverts,"binderStiffness",(T)1.0);

            ktris.resize(kb_tris.size());
            TILEVEC_OPS::copy<3>(cudaPol,kb_tris,"nrm",ktris,"nrm");
            TILEVEC_OPS::copy<3>(cudaPol,kb_tris,"inds",ktris,"inds");            
        }
        // std::cout << "nm_kb_tris : " << kb_tris.size() << " nm_kb_verts : " << kb_verts.size() << std::endl;
        // cudaPol(zs::range(kb_tris.size()),
        //     [kb_verts = proxy<space>({},kb_verts),kb_tris = proxy<space>({},kb_tris),kverts = proxy<space>({},kverts)] ZS_LAMBDA(int ti) mutable {
        //         auto tri = kb_tris.pack(dim_c<3>,"inds",ti).reinterpret_bits(int_c);
        //         for(int i = 0;i != 3;++i)
        //             atomic_add(exec_cuda,&kverts("area",tri[i]),(T)kb_tris("area",ti)/(T)3.0);
        //         if(ti == 0)
        //             printf("tri[0] area : %f\n",(float)kb_tris("area",ti));
        // });

        // the temp buffer only store the data that will change every iterations or every frame
        dtiles_t vtemp{verts.get_allocator(),
                            {
                                {"grad", 3},
                                {"P", 9},
                                {"bou_tag",1},
                                {"dir", 3},
                                {"xn", 3},
                                {"xp",3},
                                {"vn",3},
                                {"vp",3},
                                {"is_inverted",1},
                                {"active",1},
                                {"k_active",1},
                                {"ring_mask",1},
                                {"color_mask",1},
                                {"type_mask",1},
                                {"grad",3},
                                {"H",9},
                                {"inds",1}
                            },verts.size()};
        

        // auto max_collision_pairs = tris.size() / 10; 
        dtiles_t etemp(eles.get_allocator(), {
                // {"H", 12 * 12},
                    {"ActInv",3*3},
                // {"muscle_ID",1},
                    {"is_inverted",1}
                }, eles.size()
        );

                // {{tags}, cnt, memsrc_e::um, 0}
        dtiles_t sttemp(tris.get_allocator(),
            {
                {"nrm",3},
                {"inds",3},
                {"grad",9},
                {"H",9 * 9}
            },tris.size()
        );
        TILEVEC_OPS::copy(cudaPol,tris,"inds",sttemp,"inds");
        dtiles_t setemp(lines.get_allocator(),
            {
                {"nrm",3}
            },lines.size()
        );

        // std::cout << "sttemp.size() << " << sttemp.size() << std::endl;
        // std::cout << "setemp.size() << " << setemp.size() << std::endl;

        bool turn_on_self_collision = get_input2<bool>("use_self_collision");

        // int fp_buffer_size = turn_on_self_collision ? points.size() * MAX_FP_COLLISION_PAIRS : 0;

        #ifdef USE_SPARSE_MATRIX

        dtiles_t fp_buffer(points.get_allocator(),{
            {"inds",4},
            {"grad",12},
            {"H",12 * 12},
        },points.size());

        #else

        int fp_buffer_size = points.size() * MAX_FP_COLLISION_PAIRS;
        // int fp_buffer_size = 0;

        dtiles_t fp_buffer(points.get_allocator(),{
            {"inds",4},
            {"area",1},
            {"inverted",1},
        },fp_buffer_size);

        #endif

        // static dtiles_t ee_buffer(lines.get_allocator(),{
        //     {"inds",4},
        //     {"area",1},
        //     {"inverted",1},
        //     {"abary",2},
        //     {"bbary",2},
        //     {"bary",4}
        // },lines.size());

        // int ee_buffer_size = ee_buffer.size();
        int ee_buffer_size = 0;


        int kc_buffer_size = kverts.size() * MAX_FP_COLLISION_PAIRS;
        // kc_buffer_size = 0;

        dtiles_t kc_buffer(points.get_allocator(),{
            {"inds",2},
            {"area",1},
            {"inverted",1},
        },kc_buffer_size);

        // int kc_buffer_size = kc_buffer.size();
        // int kc_buffer_size = 0;

// change
#ifdef USE_SPARSE_MATRIX
        dtiles_t gh_buffer(eles.get_allocator(),{
            {"inds",4},
            {"H",12*12},
            {"grad",12}
        },eles.size());
#else

        dtiles_t gh_buffer(eles.get_allocator(),{
            {"inds",4},
            {"H",12*12},
            {"grad",12}
        },eles.size() + fp_buffer.size());
#endif

        // dtiles_t tri_gh_buffer(tris.size(),{
        //     {"inds",3},
        //     {"H",9 * 9},
        //     {"grad",9}
        // },tris.size());


        // TILEVEC_OPS::fill<4>(cudaPol,etemp,"inds",zs::vec<int,4>::uniform(-1).template reinterpret_bits<T>())
        // TILEVEC_OPS::copy<4>(cudaPol,eles,"inds",etemp,"inds");
        TILEVEC_OPS::fill<9>(cudaPol,etemp,"ActInv",zs::vec<T,9>{1.0,0.0,0.0,0.0,1.0,0.0,0.0,0.0,1.0});
        // TILEVEC_OPS::fill(cudaPol,vtemp,"inertia",(T)1.0);
        // if(verts.hasProperty("inertia"))
        //     TILEVEC_OPS::copy(cudaPol,verts,"inertia",vtemp,"inertia");
        if(verts.hasProperty("is_inverted"))
            TILEVEC_OPS::copy(cudaPol,verts,"is_inverted",vtemp,"is_inverted");
        else
            TILEVEC_OPS::fill(cudaPol,vtemp,"is_inverted",(T)0.0);
        cudaPol(zs::range(vtemp.size()),
            [vtemp = proxy<space>({},vtemp)] ZS_LAMBDA(int vi) mutable {
                vtemp("inds",vi) = reinterpret_bits<T>(vi);
        });
        // apply muscle activation

        if(!eles.hasProperty("Act"))
            eles.append_channels(cudaPol,{{"Act",1}});

        if(!eles.hasProperty(muscle_id_tag) || !eles.hasProperty("fiber"))
            fmt::print(fg(fmt::color::red),"the quadrature has no \"{}\" muscle_id_tag\n",muscle_id_tag);
        if(nm_acts == 0)
            fmt::print(fg(fmt::color::red),"no activation input\n");

        cudaPol(zs::range(eles.size()),
            [etemp = proxy<space>({},etemp),eles = proxy<space>({},eles),
                act_buffer = proxy<space>({},act_buffer),muscle_id_tag = SmallString(muscle_id_tag),nm_acts] ZS_LAMBDA(int ei) mutable {
                // auto act = eles.template pack<3>("act",ei);
                // auto fiber = etemp.template pack<3>("fiber",ei);

                vec3 act{1.0,1.0,1.0};
                vec3 fiber{};
                // float a = 1.0f;
                if(eles.hasProperty("fiber") && eles.hasProperty(muscle_id_tag) && nm_acts > 0 && (int)eles(muscle_id_tag,ei) >= 0 && fabs(eles.template pack<3>("fiber",ei).norm() - 1.0) < 0.001 && (int)eles(muscle_id_tag,ei) < act_buffer.size()){
                    fiber = eles.template pack<3>("fiber",ei);
                    auto ID = (int)eles(muscle_id_tag,ei);
                    auto a = 1. - act_buffer("act",0,ID);
                    auto b = 1. - act_buffer("act",1,ID);
                    // act = vec3{zs::sqrt(a),zs::sqrt(1./a),zs::sqrt(1./a)};
                    // auto aclamp = 
                    // act = vec3{a < 0.7 ? 0.7 : a,zs::sqrt(1./a),zs::sqrt(1./a)};
                    act = vec3{a,zs::sqrt(1./b),zs::sqrt(1./b)};
                    eles("Act",ei) = act_buffer("act",0,ID) + 1e-6;
                }else{
                    fiber = zs::vec<T,3>(1.0,0.0,0.0);
                    act = vec3{1,1,1};
                    eles("Act",ei) = (T)0.0;
                }
                if(fabs(fiber.norm() - 1.0) > 0.1) {
                    printf("invalid fiber[%d] detected : %f %f %f\n",(int)ei,
                        (float)fiber[0],(float)fiber[1],(float)fiber[2]);
                }

                vec3 dir[3];
                dir[0] = fiber;
                auto tmp = vec3{0.0,1.0,0.0};
                dir[1] = dir[0].cross(tmp);
                if(dir[1].length() < 1e-3) {
                    tmp = vec3{0.0,0.0,1.0};
                    dir[1] = dir[0].cross(tmp);
                }

                dir[1] = dir[1] / dir[1].length();
                dir[2] = dir[0].cross(dir[1]);
                dir[2] = dir[2] / dir[2].length();

                auto R = mat3{};
                for(int i = 0;i < 3;++i)
                    for(int j = 0;j < 3;++j)
                        R(i,j) = dir[j][i];

                auto Act = mat3::zeros();
                Act(0,0) = act[0];
                Act(1,1) = act[1];
                Act(2,2) = act[2];

                Act = R * Act * R.transpose();
                etemp.template tuple<9>("ActInv",ei) = zs::inverse(Act);
                // if(a < 1.0f) {
                //     auto ActInv = etemp.template pack<3,3>("ActInv",ei);
                //     printf("ActInv[%d] : \n%f %f %f\n%f %f %f\n%f %f %f\n",ei,
                //         (float)ActInv(0,0),(float)ActInv(0,1),(float)ActInv(0,2),
                //         (float)ActInv(1,0),(float)ActInv(1,1),(float)ActInv(1,2),
                //         (float)ActInv(2,0),(float)ActInv(2,1),(float)ActInv(2,2));
                // }
        });
        auto collisionStiffness = get_input2<float>("cstiffness");
        auto kineCollisionStiffness = get_input2<float>("kineCstiffness");


        // auto inset_ratio = get_input2<float>("collision_inset");
        // auto outset_ratio = get_input2<float>("collision_outset");    

        auto in_collisionEps = get_input2<float>("in_collisionEps");
        auto out_collisionEps = get_input2<float>("out_collisionEps");

        auto kine_in_collisionEps = get_input2<float>("kine_inCollisionEps");
        auto kine_out_collisionEps = get_input2<float>("kine_outCollisionEps");

        auto aniso_strength = get_input2<float>("aniso_strength");

        auto binderStiffness = get_input2<float>("binderStiffness");
        auto binderTag = get_param<std::string>("binderTag");
        auto binderThicknessTag = get_param<std::string>("binderThicknessTag");
        auto binderInversionTag = get_param<std::string>("binderInversionTag");

        auto planeConsPosTag = get_param<std::string>("planeConsPosTag");
        auto planeConsNrmTag = get_param<std::string>("planeConsNrmTag");
        auto planeConsIDTag = get_param<std::string>("planeConsIDTag");
        auto planeConsBaryTag = get_param<std::string>("planeConsBaryTag");

        auto planeConsStiffness = get_input2<float>("planeConsStiffness");

        FEMDynamicSteppingSystem A{
            verts,eles,
            points,lines,tris,
            (T)in_collisionEps,(T)out_collisionEps,
            bbw,bverts,bone_driven_weight,
            volf,dt,collisionStiffness,
            (T)kine_in_collisionEps,(T)kine_out_collisionEps,
            (T)kineCollisionStiffness,(T)aniso_strength,(T)binderStiffness,(T)planeConsStiffness};

        // std::cout << "set initial guess" << std::endl;
        // setup initial guess
        // if(verts.hasProperty("dt")) {
        //     std::cout << "verts has property 'dt'" << std::endl;
        // }

        TILEVEC_OPS::copy<3>(cudaPol,verts,"x",vtemp,"xp");
        TILEVEC_OPS::copy<3>(cudaPol,verts,"v",vtemp,"vp");
        if(verts.hasProperty("active"))
            TILEVEC_OPS::copy(cudaPol,verts,"active",vtemp,"active");
        else
            TILEVEC_OPS::fill(cudaPol,vtemp,"active",(T)1.0);

        if(verts.hasProperty("k_active"))
            TILEVEC_OPS::copy(cudaPol,verts,"k_active",vtemp,"k_active");
        else
            TILEVEC_OPS::fill(cudaPol,vtemp,"k_active",(T)1.0);

        // if there is no init_x as guess, then use the baraff witkin approach
        // if(verts.hasProperty("init_x"))
        //     TILEVEC_OPS::copy<3>(cudaPol,verts,"init_x",vtemp,"xn");   
        // else {
            // TILEVEC_OPS::add<3>(cudaPol,vtemp,"xp",1.0,"vp",dt,"xn");
        TILEVEC_OPS::copy(cudaPol,verts,"v",vtemp,"vn");  
        TILEVEC_OPS::copy(cudaPol,verts,"x",vtemp,"xn");
            // TILEVEC_OPS::add<3>(cudaPol,verts,"x",1.0,"vp",(T)0.0,"xn");  
        // }
        if(verts.hasProperty("bou_tag") && verts.getPropertySize("bou_tag") == 1)
            TILEVEC_OPS::copy(cudaPol,verts,"bou_tag",vtemp,"bou_tag");
        else
            TILEVEC_OPS::fill(cudaPol,vtemp,"bou_tag",(T)0.0);

        int max_newton_iterations = get_input2<int>("max_newton_iters");
        int nm_iters = 0;
        // make sure, at least one baraf simi-implicit step will be taken
        auto res0 = 1e10;

        auto kd_alpha = get_input2<float>("kd_alpha");
        auto kd_beta = get_input2<float>("kd_beta");
        auto kd_theta = get_input2<float>("kd_theta");

        auto max_cg_iters = get_param<int>("max_cg_iters");

        bool use_plane_constraint = get_input2<bool>("use_plane_constraint");
        bool use_binder_constraint = get_input2<bool>("use_binder_constraint");

        bool use_line_search = get_param<bool>("use_line_search");

        zs::CppTimer timer;

        #ifdef USE_SPARSE_MATRIX

        timer.tick();


        spmat_t spmat{};
        zs::Vector<int> is{verts.get_allocator(),verts.size()};
        zs::Vector<int> js{verts.get_allocator(),verts.size()};
        // init diagonal entries
        // cudaPol(zs::range(verts.size()),
        //         [is = proxy<space>(is),js = proxy<space>(js)] ZS_LAMBDA(int vi) mutable {
        //     is[vi] = js[vi] = vi;
        // });
        cudaPol(enumerate(is, js), [] ZS_LAMBDA(int no, int &i, int &j) mutable { i = j = no; });
        auto reserveStorage = [&is, &js](std::size_t n) {
            auto size = is.size();
            is.resize(size + n);
            js.resize(size + n);
            return size;
        };

        // init tet incidents' entries, off-diagonal
        auto tets_entry_offset = reserveStorage(eles.size() * 6);
        cudaPol(zs::range(eles.size()),[offset = tets_entry_offset,
                stride = eles.size(),
                is = proxy<space>(is),
                js = proxy<space>(js),
                eles = proxy<space>({},eles)] ZS_LAMBDA(int ei) mutable {
            auto inds = eles.pack(dim_c<4>,"inds",ei,int_c);
            for (int d = 1; d < 4; ++d)
                for (int k = 0; k < 4 - d; ++k)
                    if (inds[k] > inds[k + 1]) {
                        auto t = inds[k];
                        inds[k] = inds[k + 1];
                        inds[k + 1] = t;
                    }

            // <0, 1>, <0, 2>, <0, 3>, <1, 2>, <1, 3>, <2, 3>
            is[offset + ei] = inds[0];
            is[offset + stride + ei] = inds[0];
            is[offset + stride * 2 + ei] = inds[0];
            is[offset + stride * 3 + ei] = inds[1];
            is[offset + stride * 4 + ei] = inds[1];
            is[offset + stride * 5 + ei] = inds[2];

            js[offset + ei] = inds[1];
            js[offset + stride + ei] = inds[2];
            js[offset + stride * 2 + ei] = inds[3];
            js[offset + stride * 3 + ei] = inds[2];
            js[offset + stride * 4 + ei] = inds[3];
            js[offset + stride * 5 + ei] = inds[3];
        });

        spmat = spmat_t{verts.get_allocator(),(int)verts.size(),(int)verts.size()};
        spmat.build(cudaPol,(int)verts.size(),(int)verts.size(),zs::range(is),zs::range(js),zs::false_c);
        spmat.localOrdering(cudaPol, zs::false_c);
        spmat._vals.resize(spmat.nnz());
        spmat._vals.reset(0);   

        timer.tock("setup spmat");

        #endif

        auto cnorm = compute_average_edge_length(cudaPol,kverts,"x",ktris);

        auto use_sticky_condition = get_input2<bool>("use_sticky_condition");

        zs::Vector<zs::vec<int,4>> csPT{points.get_allocator(),points.size()};
        int nm_csPT = 0;

        while(nm_iters < max_newton_iterations) {
            // break;
            T e0 = (T)0;
            if(use_line_search){
                match([&](auto &elasticModel){
                    A.computeElasticBonesEnergy(cudaPol, elasticModel,vtemp,etemp,e0);
                },[](...) {
                    throw std::runtime_error("unsupported anisotropic elasticity model");
                })(models.getElasticModel());      
            }      

            TILEVEC_OPS::fill(cudaPol,gh_buffer,"grad",(T)0.0);
            TILEVEC_OPS::fill(cudaPol,gh_buffer,"H",(T)0.0);  
            TILEVEC_OPS::fill<4>(cudaPol,gh_buffer,"inds",zs::vec<int,4>::uniform(-1).reinterpret_bits(float_c)); 
            TILEVEC_OPS::fill(cudaPol,vtemp,"grad",(T)0.0);
            TILEVEC_OPS::fill(cudaPol,vtemp,"H",(T)0.0);
            TILEVEC_OPS::fill(cudaPol,sttemp,"grad",(T)0.0);
            TILEVEC_OPS::fill(cudaPol,sttemp,"H",(T)0.0);

            // if(!calculate_facet_normal(cudaPol,vtemp,"xn",tris,sttemp,"nrm")){
            //     throw std::runtime_error("fail updating facet normal");
            // }  

            A.findInversion(cudaPol,vtemp,etemp);  

            // match([&](auto &elasticModel,auto &anisoModel) -> std::enable_if_t<zs::is_same_v<RM_CVREF_T(anisoModel),zs::AnisotropicArap<float>>> {...},[](...) {
            //     A.computeGradientAndHessian(cudaPol, elasticModel,vtemp,etemp,gh_buffer,kd_alpha,kd_beta);
            // })(models.getElasticModel(),models.getAnisoElasticModel());
            timer.tick();
            match([&](auto &elasticModel,zs::AnisotropicArap<float> &anisoModel){
                A.computeGradientAndHessian(cudaPol, elasticModel,anisoModel,vtemp,etemp,gh_buffer,kd_alpha,kd_beta);
            },[](...) {
                throw std::runtime_error("unsupported anisotropic elasticity model");
            })(models.getElasticModel(),models.getAnisoElasticModel());
            // std::cout << "computePositionConstraintGradientAndHessian : " << kverts.size() << std::endl;
            // the binder constraint gradient and hessian
            if(use_binder_constraint) {
                std::cout << "apply binder constraint " << std::endl;
                A.computePositionConstraintGradientAndHessian(cudaPol,
                    vtemp,
                    binderTag,
                    binderThicknessTag,
                    binderInversionTag,
                    kverts,
                    gh_buffer);
            }else {
                std::cout << "apply no binder constraint" << std::endl;
            }
            if(verts.hasProperty(planeConsPosTag) && verts.hasProperty(planeConsNrmTag) && verts.hasProperty(planeConsIDTag) && verts.hasProperty(planeConsBaryTag) && use_plane_constraint){
                std::cout << "apply plane constraint" << std::endl;
                // A.computePlaneConstraintGradientAndHessian(cudaPol,
                
                A.computePlaneConstraintGradientAndHessian2(cudaPol,
                    vtemp,
                    sttemp,
                    kverts,
                    ktris,
                    planeConsBaryTag,
                    planeConsIDTag,
                    vtemp,
                    sttemp,cnorm,use_sticky_condition);
            }
            else{
                std::cout << "apply no plane constraint : " << 
                    verts.hasProperty(planeConsPosTag) << "\t" << 
                    verts.hasProperty(planeConsNrmTag) << "\t" << 
                    verts.hasProperty(planeConsIDTag) << "\t" << use_plane_constraint << std::endl;
            }
            if(!calculate_facet_normal(cudaPol,vtemp,"xn",tris,sttemp,"nrm")){
                throw std::runtime_error("fail updating facet normal");
            }  


            if(turn_on_self_collision) {
                // auto nm_insts = do_
                topological_sample(cudaPol,points,vtemp,"xn",surf_verts_buffer);
                auto nm_insts = do_global_self_intersection_analysis_on_surface_mesh_info(cudaPol,
                    surf_verts_buffer,"xn",surf_tris_buffer,halfedges,inst_buffer_info,gia_res);
                TILEVEC_OPS::fill(cudaPol,vtemp,"ring_mask",zs::reinterpret_bits<T>((int)0));
                TILEVEC_OPS::fill(cudaPol,vtemp,"color_mask",zs::reinterpret_bits<T>((int)0));
                TILEVEC_OPS::fill(cudaPol,vtemp,"type_mask",zs::reinterpret_bits<T>((int)0));
                cudaPol(zs::range(gia_res.size()),[
                    gia_res = proxy<space>({},gia_res),
                    vtemp = proxy<space>({},vtemp),
                    points = proxy<space>({},points)] ZS_LAMBDA(int pi) mutable {
                        auto vi = zs::reinterpret_bits<int>(points("inds",pi));
                        vtemp("ring_mask",vi) = gia_res("ring_mask",pi);
                        vtemp("color_mask",vi) = gia_res("color_mask",pi);
                        vtemp("type_mask",vi) = gia_res("type_mask",pi);
                });


                #ifdef USE_SPARSE_MATRIX
                    COLLISION_UTILS::do_facet_point_collsion_detection_and_compute_surface_normal(
                        cudaPol,
                        vtemp,"xn",
                        points,tris,sttemp,csPT,nm_csPT,(T)in_collisionEps,(T )out_collisionEps);
                    std::cout << "nm_csPT detected : " << nm_csPT << std::endl;

                    match([&](auto &elasticModel) {
                    COLLISION_UTILS::evaluate_fp_collision_grad_and_hessian(
                        cudaPol,
                        vtemp,"xn",
                        csPT,nm_csPT,
                        fp_buffer,
                        (T)in_collisionEps,(T)out_collisionEps,
                        (T)collisionStiffness,
                        elasticModel.mu,elasticModel.lam);
                    })(models.getElasticModel());

                    // auto cHn = TILEVEC_OPS::dot<12 * 12>(cudaPol,fp_buffer,"H","H");
                    // if(std::isnan(cHn)) {
                    //     std::cout << "nan cHn detected : " << std::endl;
                    //     throw std::runtime_error("nan cHn detected");
                    // }
                #else
                    match([&](auto &elasticModel) {
                        A.computeCollisionGradientAndHessian(cudaPol,elasticModel,
                            vtemp,
                            etemp,
                            sttemp,
                            setemp,
                            // ee_buffer,
                            fp_buffer,
                            kverts,
                            kc_buffer,
                            gh_buffer,kd_theta);
                        })(models.getElasticModel());
                #endif
            }

            timer.tock("eval hessian and gradient");
            timer.tick();
            // TILEVEC_OPS::fill(cudaPol,vtemp,"grad",(T)0.0); 
            TILEVEC_OPS::assemble(cudaPol,gh_buffer,"grad","inds",vtemp,"grad");
            TILEVEC_OPS::assemble(cudaPol,sttemp,"grad","inds",vtemp,"grad");

            #ifdef USE_SPARSE_MATRIX
            if(turn_on_self_collision)
                TILEVEC_OPS::assemble(cudaPol,fp_buffer,"grad","inds",vtemp,"grad");
            #endif
            TILEVEC_OPS::fill(cudaPol,vtemp,"P",(T)0.0);

            PCG::prepare_block_diagonal_preconditioner<4,3>(cudaPol,"H",gh_buffer,"P",vtemp,false,true);
            #ifdef USE_SPARSE_MATRIX
            if(turn_on_self_collision)
                PCG::prepare_block_diagonal_preconditioner<4,3>(cudaPol,"H",fp_buffer,"P",vtemp,false,true);
            #endif
            PCG::prepare_block_diagonal_preconditioner<3,3>(cudaPol,"H",sttemp,"P",vtemp,false,true);
            PCG::prepare_block_diagonal_preconditioner<1,3>(cudaPol,"H",vtemp,"P",vtemp,true,true);
            timer.tock("precondition and assemble setup");

            // eval sparse matrix
            #ifdef USE_SPARSE_MATRIX
            timer.tick();
            spmat._vals.reset(0);  

            cudaPol(zs::range(eles.size()),
                [gh_buffer = proxy<space>({},gh_buffer),
                        spmat = view<space>(spmat),
                        verts = proxy<space>({},verts)] ZS_LAMBDA(int ei) mutable {
                    auto inds = gh_buffer.pack(dim_c<4>,"inds",ei).reinterpret_bits(int_c);
                    auto H = gh_buffer.pack(dim_c<12,12>,"H",ei);
                    update_hessian(spmat,inds,H,true);
            });

             cudaPol(zs::range(sttemp.size()),
                [sttemp = proxy<space>({},sttemp),spmat = proxy<space>(spmat)] ZS_LAMBDA(int vi) mutable {
                    auto inds = sttemp.pack(dim_c<3>,"inds",vi,int_c);
                    auto H = sttemp.pack(dim_c<9,9>,"H",vi);
                    update_hessian(spmat,inds,H,true);
            });

            cudaPol(zs::range(vtemp.size()),
                [vtemp = proxy<space>({},vtemp),spmat = proxy<space>(spmat)] ZS_LAMBDA(int vi) mutable {
                    auto inds = vtemp.pack(dim_c<1>,"inds",vi,int_c);
                    auto H = vtemp.pack(dim_c<3,3>,"H",vi);
                    update_hessian(spmat,inds,H,true);
            });

            timer.tock("spmat evaluation");
            #endif
            // PCG::precondition<3>(cudaPol,vtemp,"P","grad","q");
            // T res = TILEVEC_OPS::inf_norm<3>(cudaPol, vtemp, "q");
            // if(res < newton_res){
            //     fmt::print(fg(fmt::color::cyan),"reach desire newton res {} : {}\n",newton_res,res);
            //     break;
            // }
            // auto nP = TILEVEC_OPS::inf_norm<9>(cudaPol,vtemp,"P");
            // std::cout << "nP : " << nP << std::endl;
            // PCG::prepare_block_diagonal_preconditioner<4,3>(cudaPol,"H",etemp,"P",vtemp);
            // if the grad is too small, return the result
            // Solve equation using PCG
            timer.tick();
            TILEVEC_OPS::fill(cudaPol,vtemp,"dir",(T)0.0);
            // std::cout << "solve using pcg" << std::endl;
            // auto Hn = TILEVEC_OPS::dot<12 * 12>(cudaPol,gh_buffer,"H","H");
            // std::cout << "Hn : " << Hn << std::endl;
            int nm_CG_iters = 0;
            #ifdef USE_SPARSE_MATRIX
                if(turn_on_self_collision)
                    nm_CG_iters = PCG::pcg_with_fixed_sol_solve<3>(cudaPol,vtemp,spmat,fp_buffer,"dir","bou_tag","grad","P","inds","H",(T)cg_res,max_cg_iters,100);
                else
                    nm_CG_iters = PCG::pcg_with_fixed_sol_solve<3>(cudaPol,vtemp,spmat,"dir","bou_tag","grad","P","inds","H",(T)cg_res,max_cg_iters,100);

            #else
                nm_CG_iters = PCG::pcg_with_fixed_sol_solve<3,4>(cudaPol,vtemp,gh_buffer,"dir","bou_tag","grad","P","inds","H",(T)cg_res,max_cg_iters,100);
            #endif
            timer.tock("CG SOLVER");
            fmt::print(fg(fmt::color::cyan),"nm_cg_iters : {}\n",nm_CG_iters);
            // T alpha = 1.;

            // auto nxn = TILEVEC_OPS::inf_norm<3>(cudaPol,vtemp,"xn");
            // auto ndir = TILEVEC_OPS::dot<3>(cudaPol,vtemp,"dir","dir");
            // auto nP = TILEVEC_OPS::dot<9>(cudaPol,vtemp,"P","P");

            // std::cout << "vtemp's xn : " << nxn << std::endl;
            // std::cout << "vtemp's dir : " << ndir << std::endl;
            // std::cout << "vtemp's P : " << nP << std::endl;

            if(use_line_search) {
                int search_idx = 0;     
                T alpha = (T)2.0;
                T beta = (T)0.5;
                T c1 = (T)0.0001;

                auto eg0 = (T)-1.0 * TILEVEC_OPS::dot<3>(cudaPol,vtemp,"grad","dir");
                if(eg0 > 0)
                    throw std::runtime_error("invalid searching direction");
                double armijo_condition;
                // int max_line_search = 5;


                do {
                    if(search_idx != 0){
                        TILEVEC_OPS::add<3>(cudaPol,vtemp,"xn",(T)1.0,"dir",-alpha,"xn"); 
                    }
                    alpha *= beta;
                    TILEVEC_OPS::add<3>(cudaPol,vtemp,"xn",(T)1.0,"dir",alpha,"xn"); 
                    T e1;
                    match([&](auto &elasticModel){
                        A.computeElasticBonesEnergy(cudaPol, elasticModel,vtemp,etemp,e1);
                    },[](...) {
                        throw std::runtime_error("unsupported anisotropic elasticity model");
                    })(models.getElasticModel());

                    ++search_idx;            

                    armijo_condition = double(e1) - double(e0) - double(c1) * double(alpha) * double(eg0);   
                }while(armijo_condition > 0.0);

            }else{
                cudaPol(zs::range(vtemp.size()), [vtemp = proxy<space>({}, vtemp),dt] __device__(int i) mutable {
                    vtemp.template tuple<3>("xn", i) =
                        vtemp.template pack<3>("xn", i) + vtemp.template pack<3>("dir", i);
                    vtemp.template tuple<3>("vn",i) = 
                        (vtemp.template pack<3>("xn",i) - vtemp.template pack<3>("xp",i))/dt; 
                });
            }


            // nxn = TILEVEC_OPS::inf_norm<3>(cudaPol,vtemp,"xn");
            // std::cout << "new vtemp's xn : " << nxn << std::endl;


            // res = TILEVEC_OPS::inf_norm<3>(cudaPol, vtemp, "dir");// this norm is independent of descriterization
            // std::cout << "res[" << nm_iters << "] : " << res << std::endl;
            // if(res < newton_res){
            //     fmt::print(fg(fmt::color::cyan),"reach desire newton res {} : {}\n",newton_res,res);
            //     break;
            // }
            nm_iters++;
        }


        cudaPol(zs::range(verts.size()),
                [vtemp = proxy<space>({}, vtemp), verts = proxy<space>({}, verts),dt = dt] __device__(int vi) mutable {
                    // auto newX = vtemp.pack(dim_c<3>,"xn", vi);
                    verts.tuple<3>("x", vi) = vtemp.pack(dim_c<3>,"xn", vi);
                    // if(verts.hasProperty("dt"))
                    //     dt = verts("dt",vi);
                    verts.tuple<3>("v",vi) = vtemp.pack<3>("vn",vi);
                });

        set_output("ZSParticles", zsparticles);
    }
};

ZENDEFNODE(FleshDynamicStepping, {{"ZSParticles","kinematic_boundary",
                                    "gravity","Acts",
                                    "driven_boudary",
                                    {"int","max_newton_iters","5"},
                                    {"float","cg_res","0.0001"},
                                    {"string","driven_tag","bone_bw"},
                                    {"float","driven_weight","0.02"},
                                    {"string","muscle_id_tag","ms_id_tag"},
                                    {"float","cstiffness","0.0"},
                                    {"float","in_collisionEps","0.01"},
                                    {"float","out_collisionEps","0.01"},
                                    {"float","kineCstiffness","1"},
                                    {"float","kine_inCollisionEps","0.01"},
                                    {"float","kine_outCollisionEps","0.02"},
                                    {"float","dt","0.5"},
                                    {"float","newton_res","0.001"},
                                    {"float","kd_alpha","0.01"},
                                    {"float","kd_beta","0.01"},
                                    {"float","kd_theta","0.01"},
                                    {"float","aniso_strength","1.0"},
                                    {"float","binderStiffness","1.0"},
                                    {"float","planeConsStiffness","0.01"},
                                    {"bool","use_plane_constraint","0"},
                                    {"bool","use_binder_constraint","0"},
                                    {"bool","use_self_collision","0"},
                                    {"bool","use_sticky_condition","0"}
                                    },
                                  {"ZSParticles"},
                                  {
                                    {"int","max_cg_iters","1000"}, 
                                    {"string","binderTag","binder_tag"},
                                    {"string","binderThicknessTag","binder_thickness"},
                                    {"string","binderInversionTag","binder_inversion"},
                                    {"string","planeConsPosTag","planeConsPosTag"},
                                    {"string","planeConsNrmTag","planeConsNrmTag"},
                                    {"string","planeConsIDTag","planeConsIDTag"},
                                    {"string","planeConsBaryTag","planeConsBaryTag"},
                                    {"bool","use_line_search","0"}
                                  },
                                  {"FEM"}});

// struct EvaluateElasticForce : zeno::INode {
//     using T = float;
//     using Ti = int;
//     using dtiles_t = zs::TileVector<T,32>;
//     using tiles_t = typename ZenoParticles::particles_t;
//     using vec2 = zs::vec<T,2>;
//     using vec3 = zs::vec<T, 3>;
//     using mat3 = zs::vec<T, 3, 3>;
//     using mat9 = zs::vec<T,9,9>;
//     using mat12 = zs::vec<T,12,12>;

//     using bvh_t = zs::LBvh<3,int,T>;
//     using bv_t = zs::AABBBox<3, T>;

//     using pair3_t = zs::vec<Ti,3>;
//     using pair4_t = zs::vec<Ti,4>;

//     virtual void apply() override {
//         using namespace zs;
//         auto zsparticles = get_input<ZenoParticles>("ZSParticles");
//         auto models = zsparticles->getModel();
//         auto& verts = zsparticles->getParticles();
//         auto& eles = zsparticles->getQuadraturePoints();

//         std::vector<zeno::vec2f> act_;    
//         std::size_t nm_acts = 0;

//         if(has_input("Acts")) {
//             act_ = get_input<zeno::ListObject>("Acts")->getLiterial<zeno::vec2f>();
//             nm_acts = act_.size();
//         }

//         std::cout << "nmActs:" << nm_acts << std::endl;

//         constexpr auto host_space = zs::execspace_e::openmp;
//         auto ompExec = zs::omp_exec();
//         auto act_buffer = dtiles_t{{{"act",2}},nm_acts,zs::memsrc_e::host};
//         ompExec(zs::range(act_buffer.size()),
//             [act_buffer = proxy<host_space>({},act_buffer),act_] (int i) mutable {
//                 act_buffer.tuple(dim_c<2>,"act",i) = vec2(act_[i][0],act_[i][1]);
//         });

//         act_buffer = act_buffer.clone({zs::memsrc_e::device, 0});
//         constexpr auto space = execspace_e::cuda;
//         auto cudaPol = cuda_exec();

//         auto forceTag = get_param<std::string>("forceTag");
//         if(!verts.hasProperty(forceTag)){
//             verts.append_channels(cudaPol,{{forceTag,3}});
//         }
//         TILEVEC_OPS::fill(cudaPol,verts,forceTag,(T)0.0);


//     }
// };

// struct VisualizeBoneDrivenForce : zeno::INode {

// };

// struct VisualizePlaneConstraintForce : zeno::INode {
//     using T = float;
//     using Ti = int;
//     using dtiles_t = zs::TileVector<T,32>;
//     using tiles_t = typename ZenoParticles::particles_t;
//     using vec2 = zs::vec<T,2>;
//     using vec3 = zs::vec<T, 3>;
//     using mat3 = zs::vec<T, 3, 3>;
//     using mat9 = zs::vec<T,9,9>;
//     using mat12 = zs::vec<T,12,12>;

//     using bvh_t = zs::LBvh<3,int,T>;
//     using bv_t = zs::AABBBox<3, T>;

//     using pair3_t = zs::vec<Ti,3>;
//     using pair4_t = zs::vec<Ti,4>;

//     virtual void apply() override {
//         using namespace zs;
//         auto zsparticles = get_input<ZenoParticles>("ZSParticles");
//         auto& verts = zsparticles->getParticles();
//         auto& tris  = (*zsparticles)[ZenoParticles::s_surfTriTag];     

//         auto kinematic_boundary = get_input<ZenoParticles>("kinematic_boundary");   
//         auto& kb_verts = kinematic_boundary->getParticles();
//         auto& kb_tris = kinematic_boundary->getQuadraturePoints();

//         auto planeConsPosTag = get_param<std::string>("planeConsPosTag");
//         auto planeConsNrmTag = get_param<std::string>("planeConsNrmTag");
//         auto planeConsIDTag = get_param<std::string>("planeConsIDTag");
//         auto planeConsBaryTag = get_param<std::string>("planeConsBaryTag");

//         auto planeConsStiffness = get_input2<float>("planeConsStiffness");        


//     }
// }

};